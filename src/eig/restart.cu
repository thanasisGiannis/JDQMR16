#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdio.h>
#include <assert.h>

#include "restart.h"


#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void restart_init(double *W, int ldW, double *H, int ldH, 
               double *Vprev, int ldVprev, double *Lprev,
               double *V, int ldV, double *L,
               int *basisSize, int maxBasisSize, int numEvals, int dim, 
               struct jdqmr16Info *jd){


   struct gpuHandler *gpuH    = jd->gpuH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipsparseHandle_t   cusparseH = gpuH->cusparseH;

   struct restartSpace *spRestart = jd->spRestart;

   hipMalloc((void**)&(spRestart->VprevTV),sizeof(double)*numEvals*numEvals);
   spRestart->ldVprevTV = numEvals;



   double *d_tau;  
   int    *devInfo;
   double *d_work;

   int lwork_geqrf = 0;
   int lwork_orgqr = 0;
   int lwork = 0;
   int info_gpu = 0;   

   hipMalloc((void**)&(spRestart->d_tau), sizeof(double)*dim);
   hipMalloc((void**)&(spRestart->devInfo), sizeof(int));

   
   hipsolverDnDgeqrf_bufferSize(cusolverH,dim,numEvals,V,ldV,&lwork_geqrf);
   hipsolverDnDorgqr_bufferSize(cusolverH,dim,numEvals,numEvals,V,ldV,spRestart->d_tau,&lwork_orgqr);

   spRestart->lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
   hipMalloc((void**)&(spRestart->d_work), sizeof(double)*(spRestart->lwork));



   hipsparseSpMatDescr_t descrA;
   hipsparseDnMatDescr_t descrW;
   hipsparseDnMatDescr_t descrAW;
   hipMalloc((void**)&(spRestart->AW),sizeof(double)*dim*2*numEvals);
   spRestart->ldAW = dim;

   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spRestart->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
	hipsparseCreateDnMat(&(spRestart->descrW),dim,2*numEvals,ldW,W,HIP_R_64F,HIPSPARSE_ORDER_COL);
	hipsparseCreateDnMat(&(spRestart->descrAW),dim,2*numEvals,spRestart->ldAW,spRestart->AW,HIP_R_64F,HIPSPARSE_ORDER_COL);

   size_t bufferSize;
   double zero = 0.0;
   double one  = 1.0;
   hipsparseSpMM_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spRestart->descrA,spRestart->descrW,&zero,
                        spRestart->descrAW,HIP_R_64F,HIPSPARSE_COOMM_ALG2,&(spRestart->bufferSize));

   assert(spRestart->bufferSize>0);
	hipMalloc((void**)&(spRestart->buffer),spRestart->bufferSize);

}

void restart_destroy(struct jdqmr16Info *jd){

   struct restartSpace *spRestart = jd->spRestart;

   hipFree(spRestart->AW);
   hipFree(spRestart->VprevTV);

   hipFree(spRestart->d_tau);
   hipFree(spRestart->devInfo);
   hipFree(spRestart->d_work);

   hipFree(spRestart->buffer);
}

void restart(double *W, int ldW, double *H, int ldH, 
               double *Vprev, int ldVprev, double *Lprev,
               double *V, int ldV, double *L,
               int *basisSize, int maxBasisSize, int numEvals, int dim, 
               struct jdqmr16Info *jd){

   *basisSize = 2;

   struct gpuHandler *gpuH    = jd->gpuH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipsparseHandle_t   cusparseH = gpuH->cusparseH;

   struct restartSpace *spRestart = jd->spRestart;

   hipMemset(W,0,sizeof(double)*ldW*maxBasisSize*numEvals);
   hipMemset(H,0,sizeof(double)*ldH*maxBasisSize*numEvals);

   /* W = [Vprev V]*/
   hipMemcpy(W,Vprev,sizeof(double)*ldVprev*numEvals,hipMemcpyDeviceToDevice);
   double *VprevTV   = spRestart->VprevTV;
   int     ldVprevTV = spRestart->ldVprevTV;


   double zero = 0.0;
   double one  = 1.0;

   for(int i=0; i<numEvals; i++){
      hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                              Vprev,HIP_R_64F,ldVprev,V,HIP_R_64F,ldV,&zero,
                              VprevTV,HIP_R_64F,ldVprevTV,HIP_R_64F,
                              CUBLAS_GEMM_ALGO2);
      double minus_one = -1.0;
      hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                              Vprev,HIP_R_64F,ldVprev,VprevTV,HIP_R_64F,ldVprevTV,&one,
                              V,HIP_R_64F,ldV,HIP_R_64F,
                              CUBLAS_GEMM_ALGO2);
   }

   double *d_tau = spRestart->d_tau;  
   int    *devInfo = spRestart->devInfo;
   double *d_work = spRestart->d_work;

   int lwork_geqrf = 0;
   int lwork_orgqr = 0;
   int lwork = spRestart->lwork;
   int info_gpu = 0;   


   hipsolverDnDgeqrf(cusolverH,dim,numEvals,V,ldV,d_tau,d_work,lwork,devInfo);
   hipsolverDnDorgqr(cusolverH,dim,numEvals,numEvals,V,ldV,d_tau,d_work,lwork,devInfo);

   hipMemcpy(&W[0+numEvals*ldW],V,sizeof(double)*ldV*numEvals,hipMemcpyDeviceToDevice);


   /* H = W'*A*W */
   hipsparseSpMatDescr_t descrA = spRestart->descrA;
   hipsparseDnMatDescr_t descrW = spRestart->descrW;
   hipsparseDnMatDescr_t descrAW = spRestart->descrAW;

   double *AW = spRestart->AW;
   int ldAW = spRestart->ldAW;

   double *buffer = spRestart->buffer;

   hipsparseSpMM(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrW,&zero,descrAW,HIP_R_64F,
             HIPSPARSE_COOMM_ALG2,buffer);

   hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,2*numEvals,2*numEvals,dim,&one,
                        W,HIP_R_64F,ldW,AW,HIP_R_64F,ldAW,&zero,
                        H,HIP_R_64F,ldH,HIP_R_64F,
                        CUBLAS_GEMM_ALGO2);

//   hipblasDaxpy(cublasH,numEvals,&one,Lprev,1,H,ldH+1);


}
