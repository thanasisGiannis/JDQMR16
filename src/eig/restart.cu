#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdio.h>
#include <assert.h>

#include "restart.h"


#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void restart_init(double *W, int ldW, double *H, int ldH, 
               double *Vprev, int ldVprev, double *Lprev,
               double *V, int ldV, double *L,
               int *basisSize, int maxBasisSize, int numEvals, int dim, 
               struct jdqmr16Info *jd){


   struct gpuHandler *gpuH    = jd->gpuH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipsparseHandle_t   cusparseH = gpuH->cusparseH;

   struct restartSpace *spRestart = jd->spRestart;
/*
   hipMalloc((void**)&(spRestart->VprevTV),sizeof(double)*numEvals*numEvals);
   spRestart->ldVprevTV = numEvals;
*/
   hipMalloc((void**)&(spRestart->AW),sizeof(double)*dim*3*numEvals);
   spRestart->ldAW = dim;

   double *d_tau;  
   int    *devInfo;
   double *d_work;

   int lwork_geqrf = 0;
   int lwork_orgqr = 0;
   int lwork = 0;
   int info_gpu = 0;   

   hipMalloc((void**)&(spRestart->d_tau), sizeof(double)*dim);
   hipMalloc((void**)&(spRestart->devInfo), sizeof(int));

   
   hipsolverDnDgeqrf_bufferSize(cusolverH,dim,3*numEvals,W,ldW,&lwork_geqrf);
   hipsolverDnDorgqr_bufferSize(cusolverH,dim,3*numEvals,3*numEvals,W,ldW,spRestart->d_tau,&lwork_orgqr);

   spRestart->lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
   hipMalloc((void**)&(spRestart->d_work), sizeof(double)*(spRestart->lwork));



   hipsparseSpMatDescr_t descrA;
   hipsparseDnMatDescr_t descrW;
   hipsparseDnMatDescr_t descrAW;
   hipMalloc((void**)&(spRestart->AW),sizeof(double)*dim*3*numEvals);
   spRestart->ldAW = dim;

   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spRestart->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
	hipsparseCreateDnMat(&(spRestart->descrW),dim,3*numEvals,ldW,W,HIP_R_64F,HIPSPARSE_ORDER_COL);
	hipsparseCreateDnMat(&(spRestart->descrAW),dim,3*numEvals,spRestart->ldAW,spRestart->AW,HIP_R_64F,HIPSPARSE_ORDER_COL);

   size_t bufferSize;
   double zero = 0.0;
   double one  = 1.0;
   hipsparseSpMM_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spRestart->descrA,spRestart->descrW,&zero,
                        spRestart->descrAW,HIP_R_64F,HIPSPARSE_COOMM_ALG2,&(spRestart->bufferSize));

   assert(spRestart->bufferSize>0);
	hipMalloc((void**)&(spRestart->buffer),spRestart->bufferSize);

}

void restart_destroy(struct jdqmr16Info *jd){

   struct restartSpace *spRestart = jd->spRestart;

   hipFree(spRestart->AW);
//   hipFree(spRestart->VprevTV);

   hipFree(spRestart->d_tau);
   hipFree(spRestart->devInfo);
   hipFree(spRestart->d_work);

   hipFree(spRestart->buffer);
}

void restart(double *W, int ldW, double *H, int ldH, 
               double *Vprev, int ldVprev, double *Lprev,
               double *V, int ldV, double *L, double *AWp, int ldAWp,
               int *basisSize, int maxBasisSize, int numEvals, int dim, 
               struct jdqmr16Info *jd){


   struct gpuHandler *gpuH    = jd->gpuH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipsparseHandle_t   cusparseH = gpuH->cusparseH;

   struct restartSpace *spRestart = jd->spRestart;

   //hipMemset(W,0,sizeof(double)*ldW*maxBasisSize*numEvals);
   hipMemset(H,0,sizeof(double)*ldH*maxBasisSize*numEvals);


   /* W = [Vprev V W_1]*/
   *basisSize = 3;

   hipMemcpy(W,Vprev,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
   hipMemcpy(&W[0+numEvals*ldW],V,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
   hipMemcpy(&W[0+2*numEvals*ldW],&W[0+(maxBasisSize-1)*numEvals*ldW],sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);

   /* W = orth(W) */

   double *d_tau = spRestart->d_tau;  
   int    *devInfo = spRestart->devInfo;
   double *d_work = spRestart->d_work;
   int lwork = spRestart->lwork;

   hipsolverDnDgeqrf(cusolverH,dim,3*numEvals,W,ldW,d_tau,d_work,lwork,devInfo);
   hipsolverDnDorgqr(cusolverH,dim,3*numEvals,3*numEvals,W,ldW,d_tau,d_work,lwork,devInfo);


   /* AW = A*W */
   hipsparseSpMatDescr_t descrA = spRestart->descrA;
   hipsparseDnMatDescr_t descrW = spRestart->descrW;
   hipsparseDnMatDescr_t descrAW = spRestart->descrAW;

   double *AW = spRestart->AW;
   int ldAW = spRestart->ldAW;

   double *buffer = spRestart->buffer;

   double zero = 0.0;
   double one  = 1.0;

   hipsparseSpMM(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrW,&zero,descrAW,HIP_R_64F,
             HIPSPARSE_COOMM_ALG2,buffer);


   /* W = W'*AW */
   hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,3*numEvals,3*numEvals,dim,&one,
                        W,HIP_R_64F,ldW,AW,HIP_R_64F,ldAW,&zero,
                        H,HIP_R_64F,ldH,HIP_R_64F,
                        CUBLAS_GEMM_ALGO2);

   /* AWp = AW */
   hipMemcpy(AWp,AW,sizeof(double)*dim*3*numEvals,hipMemcpyDeviceToDevice);
   
   
}


















