#include "hip/hip_runtime.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"
#include "../matrix/double2halfMat.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "initBasis.h"
#include "eigH.h"
#include "residual.h"   
#include "locking.h"

void lock_init(double *V, int ldV, double *L, double *R, int ldR, double *normr,
            double *Qlocked, int ldQlocked, double *Llocked, double *W, int ldW, double *H, int ldH, double *AW, int ldAW, 
            int &numLocked, int &numEvals, int maxBasis, int &basisSize, int dim, double tol, struct jdqmr16Info *jd){

   int memReqD    = 0;
   int memReqI    = 0;
   size_t memReqV = 0;

   struct lockSpace *spLock = jd->spLock;

   hipMalloc((void**)&(spLock->QTV),sizeof(double)*numEvals*numEvals);
   spLock->ldQTV = numEvals;

   hipMalloc((void**)&(spLock->QTR),sizeof(double)*numEvals*numEvals);
   spLock->ldQTR = numEvals; 

   hipMalloc((void**)&(spLock->PR),sizeof(double)*dim*numEvals);
   spLock->ldPR = dim;

   spLock->Lh = (double *)malloc(sizeof(double)*numEvals);
   spLock->Llockedh = (double *)malloc(sizeof(double)*numEvals);


   memReqD += numEvals*numEvals;
   memReqD += numEvals*numEvals;
   memReqD += dim*numEvals;



   jd->gpuMemSpaceDoubleSize = max(jd->gpuMemSpaceDoubleSize,memReqD);
   jd->gpuMemSpaceIntSize    = max(jd->gpuMemSpaceIntSize,memReqI);
   jd->gpuMemSpaceVoidSize   = max(jd->gpuMemSpaceVoidSize,memReqV);

//   return;
#if 0
   hipFree(spLock->QTV);
   hipFree(spLock->QTR);
   hipFree(spLock->PR);
#endif
}

void lock_destroy(struct jdqmr16Info *jd){


   struct lockSpace *spLock = jd->spLock;

   free(spLock->Lh);
   free(spLock->Llockedh);
#if 1
   hipFree(spLock->QTV);
   hipFree(spLock->QTR);
   hipFree(spLock->PR);
#endif
}


void lock(double *V, int ldV, double *L, double *R, int ldR, double *normr,
            double *Qlocked, int ldQlocked, double *Llocked, double *W, int ldW,
            double *H, int ldH, double *AW, int ldAW, double *QH,
            int ldQH, int &numLocked, int &numEvals, int maxBasis, int &basisSize, 
            int dim, double tol, struct jdqmr16Info *jd){


   if(jd->locking == 0){
      numLocked = 0;
      for(int i=0; i<numEvals; i++){
         if(normr[i] < tol){
            numLocked ++;
         }
      }
      if(numLocked == numEvals){
         hipMemcpy(Qlocked,V,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
      }

      return;
   }
   struct lockSpace *spLock = jd->spLock;

#if 1

   double *memD = jd->gpuMemSpaceDouble;
   int    *memI = jd->gpuMemSpaceInt;
   void   *memV = jd->gpuMemSpaceVoid;

   double *QTV = memD; memD += numEvals*numEvals;
   double *QTR = memD; memD += numEvals*numEvals;
   double *PR  = memD; 

   int ldQTV =  spLock->ldQTV;// numEvals;
   int ldQTR = spLock->ldQTR; 
   int ldPR = spLock->ldPR;

#else

   double *QTV = spLock->QTV; //hipMalloc((void**)&QTV,sizeof(double)*numEvals*numEvals);
   int ldQTV =  spLock->ldQTV;// numEvals;
   double *QTR = spLock->QTR;// hipMalloc((void**)&QTR,sizeof(double)*numEvals*numEvals);
   int ldQTR = spLock->ldQTR; 
   double *PR = spLock->PR;// hipMalloc((void**)&PR,sizeof(double)*dim*numEvals);
   int ldPR = spLock->ldPR;

#endif

   /* V = V - Q*Q'*V */
   struct gpuHandler *gpuH = jd->gpuH;   
   hipblasHandle_t cublasH = gpuH->cublasH;

   int loopIters = numEvals;
   int numEvalsFound = 0;
   for(int i=0; i<numEvals; i++){
      if(normr[i] < tol){
         /* lock eigenvalues */
         hipMemcpy(&Qlocked[0+numLocked*ldQlocked],&V[0+i*ldV],dim*sizeof(double),hipMemcpyDeviceToDevice);
         hipMemcpy(&Llocked[numLocked],&L[i],sizeof(double),hipMemcpyDeviceToDevice);

         /* put random vector inside previous spot of eigenvalues */
         if(jd->locking == 1){
            hiprandGenerator_t curandH = gpuH->curandH;
            hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */
            double  mean = 0.0;
            double  stddev = max(dim,numEvals);

            hiprandGenerateNormalDouble(curandH,&V[0+i*ldV],dim,mean,stddev);

            if(numLocked >= numEvals){
               numLocked = numEvals;
               break;
            }
 
         }
         numEvalsFound++;
         numLocked++;
      }
   }

   if(numLocked >= numEvals){
      numLocked = numEvals;
      return;
   }


   if(numEvalsFound>0){
      double minus_one = -1.0;
      double zero      =  0.0;
      double one       =  1.0;
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numLocked,numEvals,dim,&one,
                              Qlocked,HIP_R_64F,ldQlocked,V,HIP_R_64F,ldV,&zero,
                              QTV,HIP_R_64F,ldQTV,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));

      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numLocked,&minus_one,
                              Qlocked,HIP_R_64F,ldQlocked,QTV,HIP_R_64F,ldQTV,&one,
                              V,HIP_R_64F,ldV,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));

      /* init basis with new V */
      initBasis(W,ldW,H,ldH,V,ldV,L, AW, ldAW, dim,maxBasis,numEvals,0,jd); 
      basisSize = 1;
      eigH(V, ldV, L, W,ldW, H, ldH, numEvals, basisSize, QH, ldQH, jd);
      //residual(R, ldR, V, ldV, L, numEvals, jd);
      residual(R, ldR, V, ldV, L, AW, ldAW, QH, ldQH, numEvals,basisSize, jd);
      return;
   }


   if(numLocked<1){
      return;
   }

   /* check residual with the complement space */
   double E = sqrt(numLocked)*tol;
   //printf("E=%e\n",E);
   /* PR = R-Q*Q*'*R */
   hipMemcpy(PR,R,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);

   double minus_one = -1.0;
   double zero      =  0.0;
   double one       =  1.0;

   CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numLocked,numEvals,dim,&one,
                           Qlocked,HIP_R_64F,ldQlocked,PR,HIP_R_64F,ldPR,&zero,
                           QTR,HIP_R_64F,ldQTR,HIP_R_64F,
                           HIPBLAS_GEMM_DEFAULT));

   hipDeviceSynchronize();
   CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numLocked,&minus_one,
                           Qlocked,HIP_R_64F,ldQlocked,QTR,HIP_R_64F,ldQTR,&one,
                           PR,HIP_R_64F,ldPR,HIP_R_64F,
                           HIPBLAS_GEMM_DEFAULT));

   double *Lh = spLock->Lh; //(double *)malloc(sizeof(double)*numEvals);
   double *Llockedh = spLock->Llockedh;//(double *)malloc(sizeof(double)*numEvals);
   hipMemcpy(Lh,L,sizeof(double)*numEvals,hipMemcpyDeviceToHost);
   hipMemcpy(Llockedh,Llocked,sizeof(double)*numLocked,hipMemcpyDeviceToHost);

   for(int i=0;i<numEvals;i++){
      if(normr[i] < E){
         double normrd;
         hipblasDnrm2(jd->gpuH->cublasH,dim,&PR[0+i*ldR], 1, &normrd);
         double vita = sqrt(normr[i]*normr[i]-normrd*normrd);
         E = sqrt(tol*tol+vita*vita);
         double gama,gamad,gamap;
         gamad = abs(Lh[i] - Llockedh[0]);
         int index = 0;
         for(int j=1;j<numLocked; j++){
            if(gamad > abs(Lh[i] - Llockedh[j])){            
               gamad = abs(Lh[i] - Llockedh[j]);
               index = j;
            }
         }

         gamap = 1e+500;
         for(int j=0; j<numLocked; j++){
            if(j == i) continue;
            if(gamap > abs(Lh[i] - Lh[j])){            
               gamap = abs(Lh[i] - Lh[j]);
               index = j;
            }
         }

         gama = min(gamap,gamad);

         if(vita > tol && normrd < tol*(gamap/gama)-(tol*tol*numLocked)/gamad){
            /* lock eigenvalues */
            printf("Hey HO!\n");
            hipMemcpy(&Qlocked[0+numLocked*ldQlocked],&V[0+i*ldV],dim*sizeof(double),hipMemcpyDeviceToDevice);
            hipMemcpy(&Llocked[numLocked],&L[i],sizeof(double),hipMemcpyDeviceToDevice);

            /* put random vector inside previous spot of eigenvalues */
            hiprandGenerator_t curandH = gpuH->curandH;
            hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */
            double  mean = 0.0;
            double  stddev = max(dim,numEvals);

//            hipMemset(V,0,dim*numEvals);
            hiprandGenerateNormalDouble(curandH,&V[0+i*ldV],dim,mean,stddev); 
            numEvalsFound++;
         
            numLocked++;
            E = sqrt(numLocked)*tol;
            break;
         }
         
      }
   }

   if(numLocked >= numEvals){
      numLocked = numEvals;
      return;
   }

//return;

   if(numEvalsFound>0){

      /* V = V - Q*Q'*V */
      double minus_one = -1.0;
      double zero      =  0.0;
      double one       =  1.0;

      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numLocked,numEvals,dim,&one,
                              Qlocked,HIP_R_64F,ldQlocked,V,HIP_R_64F,ldV,&zero,
                              QTV,HIP_R_64F,ldQTV,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numLocked,&minus_one,
                              Qlocked,HIP_R_64F,ldQlocked,QTV,HIP_R_64F,ldQTV,&one,
                              V,HIP_R_64F,ldV,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));

      /* init basis with new V */
      initBasis(W,ldW,H,ldH,V,ldV,L, AW, ldAW, dim,maxBasis,numEvals,0,jd); 
      basisSize = 1;
      eigH(V, ldV, L, W,ldW, H, ldH, numEvals, basisSize, QH, ldQH, jd);
      //residual(R, ldR, V, ldV, L, numEvals, jd);
      residual(R, ldR, V, ldV, L, AW, ldAW, QH, ldQH, numEvals,basisSize, jd);
      return;
   }
}
















