#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_fp16.h>


#include "../matrix/double2halfMat.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"

#include "innerSolver.h"
#include "sqmr.h"

void innerSolver_init(double *P, int ldP, double *R, int ldR, 
                  double *V, int ldV, double *L,
                  int numEvals, int dim, struct jdqmr16Info *jd){

   struct innerSolverSpace *spInnerSolver = jd->spInnerSolver;


   hipMalloc((void**)&(spInnerSolver->B),sizeof(double)*dim*numEvals);
   spInnerSolver->ldB = dim;
   hipMalloc((void**)&(spInnerSolver->VTB),sizeof(double)*numEvals*numEvals);
   spInnerSolver->ldVTB = numEvals;
   hipMalloc((void**)&(spInnerSolver->X),sizeof(double)*numEvals*dim);
   spInnerSolver->ldX = dim;      


   spInnerSolver->maxB       = (double*)malloc(sizeof(double));
   spInnerSolver->normIndexB = (int*)malloc(sizeof(int));


   // init sQMR 
   if(jd->useHalf == 1){
      // fp16 
      CUDA_CALL(hipMalloc((void**)&(spInnerSolver->X16),sizeof(half)*dim*numEvals));
      CUDA_CALL(hipMalloc((void**)&(spInnerSolver->B16),sizeof(half)*dim*numEvals));
      spInnerSolver->spSQmr = (struct sqmrSpace *)malloc(sizeof(struct sqmrSpace));
      sqmr_init((half*)spInnerSolver->X16, dim, (half*)spInnerSolver->B16, dim, dim, 0, jd);
   }else if(jd->useHalf == 0 ){
      //fp64      
      spInnerSolver->spSQmr = (struct sqmrSpace *)malloc(sizeof(struct sqmrSpace));
      sqmrD_init((double*)spInnerSolver->X16, dim, (double*)spInnerSolver->B16, dim, V, ldV, numEvals, dim, 0, jd);
   }else if(jd->useHalf == -1){
      CUDA_CALL(hipMalloc((void**)&(spInnerSolver->X32),sizeof(float)*dim*numEvals));
      CUDA_CALL(hipMalloc((void**)&(spInnerSolver->B32),sizeof(float)*dim*numEvals));
      spInnerSolver->spSQmr = (struct sqmrSpace *)malloc(sizeof(struct sqmrSpace));
      //sqmrD_init((double*)spInnerSolver->X32, dim, (double*)spInnerSolver->B32, dim, V, ldV, numEvals, dim, 0, jd);
      sqmrF_init((float*)spInnerSolver->X32, dim, (float*)spInnerSolver->B32, dim, dim, 0, jd);
      }else if(jd->useHalf == -2 ){
         //fp64      
         spInnerSolver->spSQmr = (struct sqmrSpace *)malloc(sizeof(struct sqmrSpace));
         sqmrD_init((double*)spInnerSolver->X16, dim, (double*)spInnerSolver->B16, dim, 0, 0, numEvals, dim, 0, jd);
      }

}

void innerSolver_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace *spInnerSolver = jd->spInnerSolver;

   
   if(jd->useHalf == 1){
      sqmr_destroy(jd);
   }else if(jd->useHalf == 0){
      sqmrD_destroy(jd);
   }else if(jd->useHalf == -1){
      sqmrF_destroy(jd);
   }

   free(spInnerSolver->spSQmr);

   if(jd->useHalf == -1){
      hipFree(spInnerSolver->X32);
      hipFree(spInnerSolver->B32);
   }else if(jd->useHalf == 1){
      hipFree(spInnerSolver->X16);
      hipFree(spInnerSolver->B16);

   }
   
   hipFree(spInnerSolver->B);
   hipFree(spInnerSolver->VTB);
   hipFree(spInnerSolver->X);

   free(spInnerSolver->normIndexB);
   free(spInnerSolver->maxB);
   
}

void innerSolver(double *P, int ldP, double *R, int ldR, double *normr,
                  double *V, int ldV, double *L,
                  int numEvals, int dim, double tol, struct jdqmr16Info *jd){
/* 
1)   Ax = (I-VV')*R/||(I-VV')*R||;
2)   P   = (I-VV')*x;

For step 1 sQMR with early stopping is used
*/

   struct gpuHandler        *gpuH          = jd->gpuH;

   hipblasHandle_t         cublasH   = gpuH->cublasH;
   struct innerSolverSpace *spInnerSolver = jd->spInnerSolver;

   double *B          = spInnerSolver->B;     int ldB = spInnerSolver->ldB;
   double *VTB        = spInnerSolver->VTB;   int ldVTB = spInnerSolver->ldVTB;
   double *X          = spInnerSolver->X;     int ldX = spInnerSolver->ldX;      
   double *maxB       = spInnerSolver->maxB;
   int    *normIndexB = spInnerSolver->normIndexB;



   CUDA_CALL(hipMemset(P,0,sizeof(double)*ldP*numEvals));
   /* B = R */
   CUDA_CALL(hipMemcpy(B,R,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice));
   /* VTR = V'*R */

   if(jd->useHalf == 1){
      /* ==== FP16 SOLVER ==== */
      /* normalize B with infinity norm */
 

      /* B = -V*VTR <=> B = R-V*V'*R */
      double minus_one = -1.0;
      double zero      =  0.0;
      double one       =  1.0;

      for(int j=0; j<numEvals; j++){
         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                                 V, HIP_R_64F,ldV,R,HIP_R_64F,ldR,
                                 &zero,VTB,HIP_R_64F,ldVTB,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
         

         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                                 V, HIP_R_64F,ldV,VTB,HIP_R_64F,ldVTB,
                                 &one,B,HIP_R_64F,ldB,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));

      }




     double alpha;
      for(int i=0; i<numEvals; i++){
         double result;
         hipblasDnrm2(cublasH, dim, &B[0+i*ldP], 1, &result);
         alpha = 1.0/(result);
         hipblasDscal(cublasH,dim,&alpha,&B[0+i*ldP],1);   
      }


      /* At this point solve the numEvals systems using mixed precision */
      half *X16 = (half*)spInnerSolver->X16;
      half *B16 = (half*)spInnerSolver->B16;

      CUDA_CALL(double2halfMat(B16, dim, B, ldB, dim, numEvals));
      for(int i=0;i<numEvals; i++){
         if(normr[i] < tol){
            hipMemcpy(&X16[0+i*dim], &B16[0+i*dim], dim*sizeof(half), hipMemcpyDeviceToDevice);
            continue;
         }
         sqmr(&X16[0+i*dim], ldX, &B16[0+i*dim], ldB, dim, 1.0, jd);
      }
      CUDA_CALL(half2doubleMat(P, ldP, X16, dim, dim, numEvals));
      /* P = X-V*V'*X */
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                              V, HIP_R_64F,ldV,P,HIP_R_64F,ldP,
                              &zero,VTB,HIP_R_64F,ldVTB,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      

      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                              V, HIP_R_64F,ldV,VTB,HIP_R_64F,ldVTB,
                              &one,P,HIP_R_64F,ldP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      return;
   }else if(jd->useHalf == 0){
      /* ==== FP64 SOLVER ==== */
      double *X_;
      double *B_;

      for(int i=0;i<numEvals; i++){
         X_ = &P[0+i*ldX];
         B_ = &B[0+i*ldB];

         if(normr[i] < tol){
            hipMemcpy(X_, B_, dim*sizeof(double), hipMemcpyDeviceToDevice);
            continue;
         }

         sqmrD(X_, dim, B_, dim, V, ldV, numEvals, dim, 1.0 , jd);
      }
      return;
   }else if(jd->useHalf == -1){
      /* ==== FP32 SOLVER ==== */
      /* B = -V*VTR <=> B = R-V*V'*R */
      double minus_one = -1.0;
      double zero      =  0.0;
      double one       =  1.0;

      for(int j=0; j<numEvals; j++){
         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                                 V, HIP_R_64F,ldV,R,HIP_R_64F,ldR,
                                 &zero,VTB,HIP_R_64F,ldVTB,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
         

         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                                 V, HIP_R_64F,ldV,VTB,HIP_R_64F,ldVTB,
                                 &one,B,HIP_R_64F,ldB,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));

      }


      /* normalize B with infinity norm */
      double alpha;
      for(int i=0; i<numEvals; i++){
         double result;
         hipblasDnrm2(cublasH, dim, &B[0+i*ldP], 1, &result);
         alpha = 1.0/(result);
         hipblasDscal(cublasH,dim,&alpha,&B[0+i*ldP],1);   
      }


      /* At this point solve the numEvals systems using mixed precision */
      float *X32 = (float*)spInnerSolver->X32;
      float *B32 = (float*)spInnerSolver->B32;

      CUDA_CALL(double2floatMat(B32, dim, B, ldB, dim, numEvals));
      for(int i=0;i<numEvals; i++){
         if(normr[i] < tol){
            hipMemcpy(&X32[0+i*dim], &B32[0+i*dim], dim*sizeof(float), hipMemcpyDeviceToDevice);
            continue;
         }

         sqmrF(&X32[0+i*dim], dim, &B32[0+i*dim], dim, dim, 1.0, jd);
      }
      CUDA_CALL(float2doubleMat(P, ldP, X32, dim, dim, numEvals));
      /* P = X-V*V'*X */
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                              V, HIP_R_64F,ldV,P,HIP_R_64F,ldR,
                              &zero,VTB,HIP_R_64F,ldVTB,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      

      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                              V, HIP_R_64F,ldV,VTB,HIP_R_64F,ldVTB,
                              &one,P,HIP_R_64F,ldP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      return;






   }else if(jd->useHalf == -2){
      /* ==== FP64 v2 SOLVER ==== */
      /* B = -V*VTR <=> B = R-V*V'*R */
      double minus_one = -1.0;
      double zero      =  0.0;
      double one       =  1.0;

      for(int j=0; j<numEvals; j++){
         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                                 V, HIP_R_64F,ldV,R,HIP_R_64F,ldR,
                                 &zero,VTB,HIP_R_64F,ldVTB,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
         

         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                                 V, HIP_R_64F,ldV,VTB,HIP_R_64F,ldVTB,
                                 &one,B,HIP_R_64F,ldB,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));

      }


      /* normalize B with infinity norm */
      double alpha;
      for(int i=0; i<numEvals; i++){
         double result;
         hipblasDnrm2(cublasH, dim, &B[0+i*ldP], 1, &result);
         alpha = 1.0/(result);
         hipblasDscal(cublasH,dim,&alpha,&B[0+i*ldP],1);   
      }

      for(int i=0;i<numEvals; i++){
         if(normr[i] < tol){
            hipMemcpy(&P[0+i*dim], &B[0+i*dim], dim*sizeof(double), hipMemcpyDeviceToDevice);
            continue;
         }
         sqmrD(&P[0+i*dim], dim, &B[0+i*dim], dim, 0,0, 1,dim, 1.0, jd);
      }
      /* P = X-V*V'*X */
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                              V, HIP_R_64F,ldV,P,HIP_R_64F,ldR,
                              &zero,VTB,HIP_R_64F,ldVTB,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      

      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                              V, HIP_R_64F,ldV,VTB,HIP_R_64F,ldVTB,
                              &one,P,HIP_R_64F,ldP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      return;






   }
}











