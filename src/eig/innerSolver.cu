#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_fp16.h>


#include "../matrix/double2halfMat.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"

#include "innerSolver.h"
#include "sqmr.h"

void applyDiagMat2Vec(double *D, double *x, int dim);
void updateScaledMatrix();


void innerSolver_init(double *X, int ldX, double *R, int ldR, 
                  double *V, int ldV, double *L,
                  int numEvals, int dim, struct jdqmr16Info *jd){

   struct innerSolverSpace *spInnerSolver = jd->spInnerSolver;

   /*
      calculate the column rows
      send them to device
      memory allocation    
   */

   
   struct jdqmr16Matrix  *matrix = jd->matrix;
   double  *A    = matrix->values;
	int     *cols = matrix->cols;
	int     *rows = matrix->rows;
   int      nnz  = matrix->nnz;
   /*
      Symmetric Matrix A 
   */
   double *norms  = (double*)malloc(sizeof(double)*dim);
   memset((void*)norms, 0,sizeof(double)*dim);
   
   for(int i=0; i<nnz; i++){
         norms[rows[i]] += A[i]*A[i];
   }

   for(int i=0; i<dim; i++){
      norms[i] = sqrt(norms[i]);
   }

   /*
      allocate memory for scaled matrix
   */

   hipMalloc((void**)&(spInnerSolver->devScaledValuesD),sizeof(double)*nnz);
   hipMalloc((void**)&(spInnerSolver->devRows),sizeof(int)*nnz);
   hipMalloc((void**)&(spInnerSolver->devCols),sizeof(int)*nnz);

   hipMalloc((void**)&(spInnerSolver->devNorms),sizeof(double)*dim);
   hipMalloc((void**)&(spInnerSolver->devInvNorms),sizeof(double)*dim);


   /* 
      
   */
   hipMemcpy(spInnerSolver->devRows,rows,sizeof(int)*nnz,hipMemcpyHostToDevice);
   hipMemcpy(spInnerSolver->devCols,cols,sizeof(int)*nnz,hipMemcpyHostToDevice);
   hipMemcpy(spInnerSolver->devScaledValuesD,A,sizeof(double)*nnz,hipMemcpyHostToDevice);

   hipMemcpy(spInnerSolver->devNorms,norms,sizeof(double)*dim,hipMemcpyHostToDevice);

   for(int i=0; i<dim; i++){
      norms[i] = 1.0/norms[i];
   }

   hipMemcpy(spInnerSolver->devInvNorms,norms,sizeof(double)*dim,hipMemcpyHostToDevice);

   free(norms);
}


__global__
void applyDiagMat2Vec_Kernel(double *D, double *x,int dim){

   int i = blockIdx.x*blockDim.x+threadIdx.x;

   if(D == NULL || x == NULL) return;
   
   if(i<dim){
      x[i] = x[i]*D[i];
   }

}


void applyDiagMat2Vec(double *D, double *x, int dim){

   applyDiagMat2Vec_Kernel<<<ceil(dim/256),1>>>(D,x,dim);
}


void innerSolver_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace *spInnerSolver = jd->spInnerSolver;
   
   hipFree((spInnerSolver->devScaledValuesD));
   hipFree((spInnerSolver->devRows));
   hipFree((spInnerSolver->devCols));

   hipFree((spInnerSolver->devNorms));
   hipFree((spInnerSolver->devInvNorms));

}

void innerSolver(double *X, int ldX, double *R, int ldR, 
                  double *V, int ldV, double *L,
                  int numEvals, int dim, struct jdqmr16Info *jd){

   struct innerSolverSpace *spInnerSolver = jd->spInnerSolver;

/*
   double *A = devScaledValuesD; //double precision
   void   *A = devScaledValuesH; //half   precision
   int    *devRows;
   int    *devCols;
*/
   double *norms    = (double*)spInnerSolver->devNorms;
   double *invNorms = (double*)spInnerSolver->devInvNorms;


   hipMemcpy(X,R,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
   
   
   /*
      first thing first to calculate the new scaled matrix
      taking account the new ritz values

      if needed cast vectors to reduced precision

      and then solve
   */
   for(int i=0;i<numEvals;i++){
      applyDiagMat2Vec(norms, &X[0+i*ldX], dim);
      applyDiagMat2Vec(invNorms, &X[0+i*ldX], dim);

   }
}
