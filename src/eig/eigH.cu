#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdio.h>
#include <assert.h>

#include "eigH.h"


#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void eigH_init(double *W, int ldW, double *L, double *H, int ldH, int numEvals, int maxBasisSize, struct jdqmr16Info *jd){


   int sizeQH = numEvals*maxBasisSize;

   struct gpuHandler *gpuH    = jd->gpuH;
   struct eigHSpace  *spEig   = jd->spEigH;
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   



   //int lwork      = spEig->lwork;   
   //int *devInfo   = spEig->devInfo; 
   hipMalloc((void**)&(spEig->devInfo),sizeof(int));
   //double *d_work = spEig->d_work;
   
   spEig->ldQH = numEvals*maxBasisSize;
   hipMalloc((void**)&(spEig->QH), sizeof(double)*(spEig->ldQH)*(numEvals*maxBasisSize));
   hipMalloc((void**)&(spEig->LH), sizeof(double)*(numEvals*maxBasisSize));


   hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
   hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

   hipsolverDnDsyevd_bufferSize(cusolverH,jobz,uplo,sizeQH,spEig->QH,spEig->ldQH,spEig->LH,&(spEig->lwork));

   hipMalloc((void**)&(spEig->d_work), sizeof(double)*(spEig->lwork));

}

void eigH_destroy(struct jdqmr16Info *jd){

   //struct gpuHandler *gpuH    = jd->gpuH;
   struct eigHSpace  *spEig   = jd->spEigH;

   hipFree(spEig->QH);
   hipFree(spEig->LH);
   hipFree(spEig->devInfo);
   hipFree(spEig->d_work);


}



void eigH(double *V, int ldV, double *L, double *W, int ldW, double *H, int ldH, int numEvals, int basisSize, struct jdqmr16Info *jd){


   int sizeQH = numEvals*basisSize;
   struct jdqmr16Matrix  *matrix = jd->matrix;
   int dim    = matrix->dim;
   struct gpuHandler *gpuH    = jd->gpuH;
   struct eigHSpace  *spEig   = jd->spEigH;
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;

   /* copy H to QH so syevd can handle the eigenvectors correctly */ 
   double *LH   = spEig->LH;
   double *QH   = spEig->QH;
   int     ldQH = spEig->ldQH;
   
   double one  = 1.0;
   double zero = 0.0;

   hipMemset(QH,0,sizeof(double)*ldQH*sizeQH);
   hipblasDgeam(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,sizeQH,sizeQH,&one,H, ldH,&zero,QH, ldQH, QH,ldQH);

   hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
   hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
   
   hipsolverDnDsyevd(cusolverH,jobz,uplo,sizeQH,QH,ldQH,LH,spEig->d_work,spEig->lwork,spEig->devInfo);
   /* eigenvalues are in a ascending order */
   /* next step to choose which part of the spectrum is needed (smallest or largest) */
   /* for starters we get smallest */
   hipMemcpy(L,LH,numEvals*sizeof(double),hipMemcpyDeviceToDevice);

   /* V = W*QH */
   hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,basisSize*numEvals,&one,
                           W,HIP_R_64F,ldW,QH,HIP_R_64F,ldQH,&zero,
                           V,HIP_R_64F,ldV,HIP_R_64F,
                           CUBLAS_GEMM_ALGO2);

}



















