#include "hip/hip_runtime.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"


#include <hiprand.h>
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "sqmr.h"

void sqmrD(double *X, int ldX, double *B, int ldB, double *V,int ldV, int numEvals,
            int dim, double infNormB, struct jdqmr16Info *jd){
/*
   Solving AX=B with sQMR and early stopping criteria

*/

   // this should be input in this function
   double ita    = 0.0;
   double thita_ = 0.0; 
   int qmrMaxIt  = 1000;
   double tol    = 1e-08;

   
   double Thita_ = 0.0;
   double rho_;
   double sigma;
   double alpha;  
   double normr;
   double Thita;
   double c;
   double g_;
   double r00;
   double rho;
   double vita;
   double g;
   double gama;
   double xi;
   double normt; 
   double f;
   double p;
   double thita;
   double pk;
   double rkm;
   double scalw;
      

   double minus_alpha;
   double deltaScal1;
   double deltaScal2;
   //double one32 = 1.0;
      
   double BITA  = 0.0;
   double DELTA = 0.0;
   double GAMA  = 0.0;
   double FI    = 0.0;
   double PSI   = 0.0;


   double minus_one = -1.0;
   double one       =  1.0;
   double zero      =  0.0;



   double *x = X;
   double *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;

   hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);
   hipDeviceSynchronize();


   double *t = x;
   double *delta = (double*)spsqmr->delta;
   double *r     = (double*)spsqmr->r;     
   double *d     = (double*)spsqmr->d;     
   double *w     = (double*)spsqmr->w;     
   double *VTd   = (double*)spsqmr->VTd;

   /* cusparse data initilization */
   //struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   //size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;


   /* r = -b */
   hipMemcpy(r,b,sizeof(double)*dim,hipMemcpyDeviceToDevice);
   hipblasScalEx(cublasH,dim,&minus_one,HIP_R_64F,r,HIP_R_64F,1,HIP_R_64F);

   /* d = r */
   hipMemcpy(d,r,sizeof(double)*dim,hipMemcpyDeviceToDevice);
   hipblasNrm2Ex(cublasH,dim,b,HIP_R_64F,1,&g,HIP_R_64F,HIP_R_64F);


   hipblasDotEx(cublasH,dim,(void*)r,HIP_R_64F,1,(void*)d,HIP_R_64F,1,(void*)&rho_,HIP_R_64F,HIP_R_64F);

   /* sQMR main iteration */
   for(int i=0; i<qmrMaxIt; i++){
      jd->innerIterations++;

      /* d = d - VVTd */
      hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,1,dim,&one,
                              V, HIP_R_64F,ldV,d,HIP_R_64F,dim,
                              &zero,VTd,HIP_R_64F,numEvals,HIP_R_64F,
                              CUBLAS_GEMM_ALGO2);
      

      hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                              V, HIP_R_64F,ldV,VTd,HIP_R_64F,numEvals,
                              &one,d,HIP_R_64F,dim,HIP_R_64F,
                              CUBLAS_GEMM_ALGO2);

      /* w = A*d */
      hipsparseSpMV(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrd,&zero,descrw,HIP_R_64F,
             HIPSPARSE_COOMV_ALG,buffer);
      jd->numMatVecsfp64++;


      /* w = w-VVTw */
      hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,1,dim,&one,
                              V, HIP_R_64F,ldV,w,HIP_R_64F,dim,
                              &zero,VTd,HIP_R_64F,numEvals,HIP_R_64F,
                              CUBLAS_GEMM_ALGO2);
      

      hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numEvals,&minus_one,
                              V, HIP_R_64F,ldV,VTd,HIP_R_64F,numEvals,
                              &one,w,HIP_R_64F,dim,HIP_R_64F,
                              CUBLAS_GEMM_ALGO2);


      /* sigma = d'*w */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,d,HIP_R_64F,1,w,HIP_R_64F,1,&sigma,HIP_R_64F,HIP_R_64F));
      /* alpha = rho_/sigma */
      alpha =1*rho_/sigma;
   

      /* r = r -alpha*w */
      minus_alpha = -1*alpha;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_alpha,HIP_R_64F,w,HIP_R_64F,1,r,HIP_R_64F,1,HIP_R_64F));

      /* Thita = norm(r)/g */
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_64F,1,&normr,HIP_R_64F,HIP_R_64F));
      Thita = normr/g;

      /* c = 1./sqrt(1+Thita*Thita) */
      c = sqrt(1/(1+Thita*Thita));
      /* g = g*Thita*c */
      g =g*Thita*c;

      if(i == 0){
         g_ = g;
      }

      /* delta = (c*c*alpha)*d + (c*c*Thita_*Thita_)*delta  */
      deltaScal1 = c*c*Thita_*Thita_;   
      deltaScal2 = c*c*alpha;
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&deltaScal1,HIP_R_64F,delta,HIP_R_64F,1,HIP_R_64F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&deltaScal2,HIP_R_64F,d,HIP_R_64F,1,delta,HIP_R_64F,1,HIP_R_64F));
 
     /*  t  = t + delta */
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_64F,delta,HIP_R_64F,1,t,HIP_R_64F,1,HIP_R_64F));

      if(abs(g) < tol){
         break;
      }

      gama = c*c*Thita_; 
      xi = c*c*alpha;    
      //normt; 
      hipblasNrm2Ex(cublasH,dim,r,HIP_R_64F,1,&normt,HIP_R_64F,HIP_R_64F); 
      f = 1 + normt*normt; 
      PSI = gama*(PSI + FI);
      FI = gama*gama*FI + xi*xi*sigma;
      GAMA = GAMA  + 2*PSI + FI;


      DELTA = gama*DELTA - xi*rho_;
      BITA = BITA + DELTA;
      p = ((thita_-ita+2*BITA+GAMA))/f;
      thita = ita+p;

      pk = (((thita_)-ita+BITA)*((thita_)-ita+BITA))/f - p*p;
      rkm = sqrt(((g)*(g))/f + pk);


      if(i==0){
         r00 = rkm;
      }


      if(rho_ < tol){
         break;
      }


      rkm = sqrt(g*g/f);

       if( (g < rkm*max(0.99 * sqrt(f),sqrt(g/g_))) || (thita > thita_) || rkm<0.1*r00  || g < tol || rkm < tol){
          break; 
       }

      /*  w = r./norm(r); */
      hipMemcpy(w,r,sizeof(double)*dim,hipMemcpyDeviceToDevice);
      hipblasNrm2Ex(cublasH,dim,w,HIP_R_64F,1,&scalw,HIP_R_64F,HIP_R_64F); 
      scalw = 1/scalw;
      hipblasScalEx(cublasH,dim,&scalw,HIP_R_64F,w,HIP_R_64F,1,HIP_R_64F);

         
      /* rho = r'*w; */
      hipblasDotEx(cublasH,dim,(void*)r,HIP_R_64F,1,(void*)w,HIP_R_64F,1,(void*)&rho,HIP_R_64F,HIP_R_64F);
      vita = rho/rho_;

      /* d = w + vita*d; */
      hipblasScalEx(cublasH,dim,&vita,HIP_R_64F,d,HIP_R_64F,1,HIP_R_64F);
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_64F,w,HIP_R_64F,1,d,HIP_R_64F,1,HIP_R_64F));


      thita_ = thita;
      Thita_ = Thita;
      rho_ = rho;
      g_ = g;

   }

}

void sqmrD_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);
   hipFree(spsqmr->VTd);

}



void sqmrD_init(double *X, int ldX, double *B, int ldB, double *V,int ldV, int numEvals,
               int dim, double infNormB, struct jdqmr16Info *jd){

   //double tmpScalar;
   //double minus_one = -1.0;
   double one       =  1.0;
   double zero      =  0.0;



   //double *x = X;
   //double *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   //hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   //double *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->VTd),sizeof(double)*numEvals);

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)spsqmr->d,HIP_R_64F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)spsqmr->w,HIP_R_64F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_64F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);



   //assert(spsqmr->bufferSize>0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);



}









void sqmr_init(half *X, int ldX, half *B, int ldB,double *V,int ldV, int numEvals,
                int dim, double infNormB, struct jdqmr16Info *jd){

 
   //half tmpScalar;
   half minus_one = __float2half(-1.0);
   half one  = __float2half(1.0);
   half zero = __float2half(0.0);



   //half *x = X;
   //half *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   //hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   //half *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(half)*dim);

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesH,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_16F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)spsqmr->d,HIP_R_16F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)spsqmr->w,HIP_R_16F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_16F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);



   //assert(spsqmr->bufferSize>=0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);


}

void sqmr_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);

}

void sqmr(half *X, int ldX,   half *B, int ldB, half *V,int ldV, int numEvals, int dim, double infNormB, struct jdqmr16Info *jd){

/*
   Solving AX=B with sQMR and early stopping criteria

*/

   // this should be input in this function
   float ita    = 0.0;
   float thita_ = 0.0; 

   int    qmrMaxIt = 1000;
   double tol      = 1e-08;

   
   float Thita_ = 0.0;
   float rho_;
   float sigma;
   float alpha;  
   float normr;
   float Thita;
   float c;
   float g_;
   float r00;
   float rho;
   float vita;
   float g;
   float gama;
   float xi;
   float normt; 
   float f;
   float p;
   float thita;
   float pk;
   float rkm;
   float scalw;
      

   float minus_alpha;
   float deltaScal1;
   float deltaScal2;
   float one32 = 1.0;
      
   float BITA  = 0.0;
   float DELTA = 0.0;
   float GAMA  = 0.0;
   float FI    = 0.0;
   float PSI   = 0.0;


   half tmpScalar;
   half minus_one = __float2half(-1.0);
   half one  = __float2half(1.0);
   half zero = __float2half(0.0);



   half *x = X;
   half *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;

   hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);
   hipDeviceSynchronize();


   half *t = x;
   half *delta = (half*)spsqmr->delta;
   half *r = (half*)spsqmr->r;     
   half *d = (half*)spsqmr->d;     
   half *w = (half*)spsqmr->w;     

   /* cusparse data initilization */
   //struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   //size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;


   /* r = -b */
   hipMemcpy(r,b,sizeof(half)*dim,hipMemcpyDeviceToDevice);
   hipblasScalEx(cublasH,dim,&minus_one,HIP_R_16F,r,HIP_R_16F,1,HIP_R_32F);

   /* d = r */
   hipMemcpy(d,r,sizeof(half)*dim,hipMemcpyDeviceToDevice);
   hipblasNrm2Ex(cublasH,dim,b,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F);
   g = __half2float(tmpScalar);


   hipblasDotEx(cublasH,dim,(void*)r,HIP_R_16F,1,(void*)d,HIP_R_16F,1,(void*)&tmpScalar,HIP_R_16F,HIP_R_32F);
   rho_ = __half2float(tmpScalar);

   /* sQMR main iteration */
   for(int i=0; i<qmrMaxIt; i++){
      jd->innerIterations++;
      
      /* w = A*d */
      hipsparseSpMV(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrd,&zero,descrw,HIP_R_16F,
             HIPSPARSE_COOMV_ALG,buffer);
      jd->numMatVecsfp16++;

      /* sigma = d'*w */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,d,HIP_R_16F,1,w,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
      sigma = __half2float(tmpScalar);
      /* alpha = rho_/sigma */
      alpha =1*rho_/sigma;
   

      /* r = r -alpha*w */
      minus_alpha = -1*alpha;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_alpha,HIP_R_32F,w,HIP_R_16F,1,r,HIP_R_16F,1,HIP_R_32F));

      /* Thita = norm(r)/g */
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
      normr = __half2float(tmpScalar);
      Thita = normr/g;

      /* c = 1./sqrt(1+Thita*Thita) */
      c = sqrt(1/(1+Thita*Thita));
      /* g = g*Thita*c */
      g =g*Thita*c;

      if(i == 0){
         g_ = g;
      }

      /* delta = (c*c*alpha)*d + (c*c*Thita_*Thita_)*delta  */
      deltaScal1 = c*c*Thita_*Thita_;   
      deltaScal2 = c*c*alpha;
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&deltaScal1,HIP_R_32F,delta,HIP_R_16F,1,HIP_R_32F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&deltaScal2,HIP_R_32F,d,HIP_R_16F,1,delta,HIP_R_16F,1,HIP_R_32F));
      /*  t  = t + delta */
      
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one32,HIP_R_32F,delta,HIP_R_16F,1,t,HIP_R_16F,1,HIP_R_32F));

      if(abs(g) < tol){
         break;
      }

      gama = c*c*Thita_; 
      xi = c*c*alpha;    
      //normt; 
      hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F); 
      normt = __half2float(tmpScalar);
      f = 1 + normt*normt; 
      PSI = gama*(PSI + FI);
      FI = gama*gama*FI + xi*xi*sigma;
      GAMA = GAMA  + 2*PSI + FI;


      DELTA = gama*DELTA - xi*rho_;
      BITA = BITA + DELTA;
      p = ((thita_-ita+2*BITA+GAMA))/f;
      thita = ita+p;

      pk = (((thita_)-ita+BITA)*((thita_)-ita+BITA))/f - p*p;
      rkm = sqrt(((g)*(g))/f + pk);


      if(i==0){
         r00 = rkm;
      }


      if(rho_ < tol){
         break;
      }


      rkm = sqrt(g*g/f);

       if( (g < rkm*max(0.99 * sqrt(f),sqrt(g/g_))) || (thita > thita_) || rkm<0.1*r00  || g < tol || rkm < tol){
          break; 
       }

      /*  w = r./norm(r); */
      hipMemcpy(w,r,sizeof(half)*dim,hipMemcpyDeviceToDevice);
      hipblasNrm2Ex(cublasH,dim,w,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F); 
      scalw = 1/__half2float(tmpScalar);
      hipblasScalEx(cublasH,dim,&scalw,HIP_R_32F,w,HIP_R_16F,1,HIP_R_32F);

         
      /* rho = r'*w; */
      hipblasDotEx(cublasH,dim,(void*)r,HIP_R_16F,1,(void*)w,HIP_R_16F,1,(void*)&tmpScalar,HIP_R_16F,HIP_R_32F);
      rho = __half2float(tmpScalar);
      vita = rho/rho_;



      /* d = w + vita*d; */
      hipblasScalEx(cublasH,dim,&vita,HIP_R_32F,d,HIP_R_16F,1,HIP_R_32F);
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one32,HIP_R_32F,w,HIP_R_16F,1,d,HIP_R_16F,1,HIP_R_32F));


      thita_ = thita;
      Thita_ = Thita;
      rho_ = rho;
      g_ = g;

   }


}
