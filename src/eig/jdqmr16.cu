#include "../../include/jdqmr16.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/helper.h"

void init_jdqmr16(struct jdqmr16Info *jd){
   
   /* allocate gpu memory */
   struct jdqmr16Matrix *A = jd->matrix;   
   
   double *vals    = A->values;
   int    *rows    = A->rows;
   int    *cols    = A->cols;

   double *devVals  = A->devValuesD;
   int    *devRows  = A->devRows;
   int    *devCols  = A->devCols;
   int     nnz      = A->nnz;
   int     dim      = A->dim;
   int     numEvals = jd->numEvals;
   int     maxBasis = jd->maxBasis;

   /* initialize data to device */
   CUDA_CALL(hipMalloc((void**)&devVals,sizeof(double)*nnz));
   CUDA_CALL(hipMalloc((void**)&devCols,sizeof(double)*nnz));
   CUDA_CALL(hipMalloc((void**)&devRows,sizeof(double)*(nnz+1)));

   CUDA_CALL(hipMemcpy((void*)devVals,(void*)vals,sizeof(double)*nnz,hipMemcpyHostToDevice));
   CUDA_CALL(hipMemcpy((void*)devCols,(void*)cols,sizeof(double)*nnz,hipMemcpyHostToDevice));
   CUDA_CALL(hipMemcpy((void*)devRows,(void*)rows,sizeof(double)*(nnz+1),hipMemcpyHostToDevice));



   /* allocate device memory for solver */
   jd->sp = (struct devSolverSpace*)malloc(sizeof(struct devSolverSpace));
   struct devSolverSpace* sp = jd->sp;
   CUDA_CALL(hipMalloc((void**)&sp->W,sizeof(double)*dim*maxBasis*numEvals));


   return;
}

void destroy_jdqmr16(struct jdqmr16Info *jd){
   /* allocate gpu memory */
   struct jdqmr16Matrix *A = jd->matrix;   
   
   double *devVals = A->devValuesD;
   int    *devRows = A->devRows;
   int    *devCols = A->devCols;
   int    nnz      = A->nnz;
   int    dim      = A->dim;

   CUDA_CALL(hipFree(devVals));
   CUDA_CALL(hipFree(devCols));
   CUDA_CALL(hipFree(devRows));


   struct devSolverSpace *sp = jd->sp;
   CUDA_CALL(hipFree(sp->W));

   free(jd->sp);


   return;

}

void jdqmr16(){

   printf("jdqmr16\n");
}

