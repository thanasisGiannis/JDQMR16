#include "hip/hip_runtime.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"
#include "../matrix/double2halfMat.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "initBasis.h"
#include "eigH.h"
#include "residual.h"   
#include "expandBasis.h"
#include "restart.h"
#include "innerSolver.h"

void init_jdqmr16(struct jdqmr16Info *jd){
   
   /* allocate gpu memory */
   struct jdqmr16Matrix *A = jd->matrix;   
   
   double *vals    = A->values;
   int    *rows    = A->rows;
   int    *cols    = A->cols;

   //double *devVals  = A->devValuesD;
   //int    *devRows  = A->devRows;
   //int    *devCols  = A->devCols;
   int     nnz      = A->nnz;
   int     dim      = A->dim;
   int     numEvals = jd->numEvals;
   int     maxBasis = jd->maxBasis;

   jd->outerIterations = 0;
   jd->innerIterations = 0;

   if(jd->useHalf !=0){
      jd->useHalf = 1;
   }
   /* if matrix is small */
   if(numEvals*maxBasis >= dim){
      jd->maxBasis = floor(dim/numEvals);
      maxBasis = jd->maxBasis;
   }
 
   /* initialize data to device */
   CUDA_CALL(hipMalloc((void**)&(A->devValuesD),sizeof(double)*nnz));
   CUDA_CALL(hipMalloc((void**)&(A->devValuesH),sizeof(half)*nnz));

   CUDA_CALL(hipMalloc((void**)&(A->devCols),sizeof(int)*nnz));
   CUDA_CALL(hipMalloc((void**)&(A->devRows),sizeof(int)*nnz));

   CUDA_CALL(hipMemcpy((void*)(A->devValuesD),(void*)vals,sizeof(double)*nnz,hipMemcpyHostToDevice));
   CUDA_CALL(hipMemcpy((void*)(A->devCols),(void*)cols,sizeof(int)*nnz,hipMemcpyHostToDevice));
   CUDA_CALL(hipMemcpy((void*)(A->devRows),(void*)rows,sizeof(int)*nnz,hipMemcpyHostToDevice));



   //CUBLAS_CALL(cublas);

   /* allocate device memory for solver */
   jd->sp   = (struct devSolverSpace*)malloc(sizeof(struct devSolverSpace));
   jd->gpuH = (struct gpuHandler*)malloc(sizeof(struct gpuHandler));
   
   /* initialize gpu handlers */
   struct gpuHandler *gpuH = jd->gpuH;   
   // hiprand
   hiprandGenerator_t *curandH = &(gpuH->curandH);
	hiprandCreateGenerator(curandH, HIPRAND_RNG_PSEUDO_DEFAULT);
   // cusolver
   hipsolverHandle_t *cusolverH = &(gpuH->cusolverH);
   hipsolverDnCreate(cusolverH);
   // cublas
   hipblasHandle_t *cublasH =  &(gpuH->cublasH);
   hipblasCreate(cublasH);   
   // cusparse
   hipsparseHandle_t *cusparseH = &(gpuH->cusparseH);
   hipsparseCreate(cusparseH);

   /* initialize space for solver */
   struct devSolverSpace* sp = jd->sp;
   CUDA_CALL(hipMalloc((void**)&sp->W,sizeof(double)*dim*maxBasis*numEvals));               sp->ldW     = dim;
   CUDA_CALL(hipMalloc((void**)&sp->H,sizeof(double)*maxBasis*numEvals*maxBasis*numEvals)); sp->ldH     = maxBasis*numEvals;
   CUDA_CALL(hipMalloc((void**)&sp->Vprev,sizeof(double)*numEvals*dim));                    sp->ldVprev = dim;
   CUDA_CALL(hipMalloc((void**)&sp->V,sizeof(double)*numEvals*dim));                        sp->ldV     = dim;
   CUDA_CALL(hipMalloc((void**)&sp->L,sizeof(double)*numEvals)); 
   CUDA_CALL(hipMalloc((void**)&sp->R,sizeof(double)*numEvals*dim));                        sp->ldR     = dim;

   CUDA_CALL(hipMalloc((void**)&sp->AW,sizeof(double)*maxBasis*numEvals*dim));              sp->ldAW    = dim;
   CUDA_CALL(hipMalloc((void**)&sp->P,sizeof(double)*numEvals*dim));                        sp->ldP     = dim;
   

   //double *H        = sp->H;        /* projected Matrix */
   //double *V        = sp->V;        /* Ritz vectors */
   //double *W        = sp->L;        /* Ritz values */
   
   /* init inner functions */

   // init initBasis
   jd->spInitBasis = (struct initBasisSpace *)malloc(sizeof(struct initBasisSpace));
   initBasis_init(sp->W,sp->ldW, sp->H, sp->ldH, sp->V, sp->ldV,sp->L, dim, maxBasis,numEvals,jd);

   // init eigH
   jd->spEigH = (struct eigHSpace *)malloc(sizeof(struct eigHSpace));   
   eigH_init(sp->W, sp->ldW, sp->L, sp->H, sp->ldH, numEvals, maxBasis, jd);
   
   // init residual
   jd->spResidual = (struct residualSpace *)malloc(sizeof(struct residualSpace));
   residual_init(sp->R,sp->ldR,sp->V,sp->ldV,sp->L,numEvals,jd);

   // init expandBasis
   jd->spExpandBasis = (struct expandBasisSpace *)malloc(sizeof(struct expandBasisSpace));
   expandBasis_init(sp->W, sp->ldW, sp->H, sp->ldH, sp->P, sp->ldP,maxBasis, dim, numEvals, jd);

   // init restart
   jd->spRestart = (struct restartSpace *)malloc(sizeof(struct restartSpace));
   restart_init(sp->W, sp->ldW, sp->H, sp->ldH, sp->Vprev, sp->ldVprev, NULL , sp->V, sp->ldV, sp->L,
               &maxBasis, maxBasis, numEvals, dim, jd);

   // init innerSolver
   jd->spInnerSolver = (struct innerSolverSpace*)malloc(sizeof(struct innerSolverSpace));
   innerSolver_init(sp->P, sp->ldP, sp->R, sp->ldR, sp->V, sp->ldV, sp->L, numEvals, dim,jd);


   /*
      find norm of matrix 
      needed for termination criteria      
   */

   jd->normMatrix = 0;
   double *val = A->values;
   for(int i=0; i<A->nnz; i++){
      if(abs(val[i]) > jd->normMatrix){
         jd->normMatrix = abs(val[i]);      
      }
   }


   return;
}

void destroy_jdqmr16(struct jdqmr16Info *jd){

   /* destroy inner functions */
   //sqrm_destroy(jd);
   innerSolver_destroy(jd);
   restart_destroy(jd);
   expandBasis_destroy(jd);
   initBasis_destroy(jd);
   eigH_destroy(jd);
   residual_destroy(jd);

   /* destroy gpu handlers */ 
   struct gpuHandler *gpuH = jd->gpuH;   

   // hiprand     
   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandDestroyGenerator(curandH);
   // cusolver
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
	hipsolverDnDestroy(cusolverH);
	// cublas
   hipblasHandle_t cublasH = gpuH->cublasH;
   hipblasDestroy(cublasH);
   // cusparse
   hipsparseHandle_t cusparseH = gpuH->cusparseH;
   hipsparseDestroy(cusparseH);

   /* Destroy Matrix */
   struct jdqmr16Matrix *A = jd->matrix;   
   
   double *devVals  = A->devValuesD;
   half   *devValsH = A->devValuesH;
   int    *devRows  = A->devRows;
   int    *devCols  = A->devCols;
   ///int    nnz       = A->nnz;
   //int    dim       = A->dim;

   CUDA_CALL(hipFree(devVals));
   CUDA_CALL(hipFree(devValsH));
   CUDA_CALL(hipFree(devCols));
   CUDA_CALL(hipFree(devRows));


   struct devSolverSpace *sp = jd->sp;


   CUDA_CALL(hipFree(sp->AW));
   CUDA_CALL(hipFree(sp->P));
   CUDA_CALL(hipFree(sp->R));
   CUDA_CALL(hipFree(sp->W));
   CUDA_CALL(hipFree(sp->H));
   CUDA_CALL(hipFree(sp->Vprev));
   CUDA_CALL(hipFree(sp->V));
   CUDA_CALL(hipFree(sp->L));

   free(jd->spInnerSolver);
   free(jd->spRestart);
   free(jd->spExpandBasis);
   free(jd->spInitBasis);
   free(jd->spEigH);
   free(jd->spResidual);

   free(jd->gpuH);
   free(jd->sp);
   
   return;

}

void jdqmr16(struct jdqmr16Info *jd){

   /* Generalized Davidson Iteration */
   struct jdqmr16Matrix *A = jd->matrix;   

   struct devSolverSpace* sp = jd->sp;
   double *W  = sp->W;  int ldW = sp->ldW; /* GD basis */
   double *H  = sp->H;  int ldH = sp->ldH; /* projected Matrix */
   double *V  = sp->V;  int ldV = sp->ldV; /* Ritz vectors */
   double *Vprev  = sp->Vprev;  int ldVprev = sp->ldVprev; /* Ritz vectors */
   
   double *L      = sp->L;                     /* Ritz values */
   
   double *R  = sp->R;  int ldR = sp->ldR; /* Ritz vectors */
   double *P  = sp->P;  int ldP = sp->ldP;
   double *AW = sp->AW; int ldAW = sp->ldAW;

   int     dim       = A->dim;         /* dimension of the problem */
   int     numEvals  = jd->numEvals;   /* number of wanted eigenvalues */
   int     maxBasis  = jd->maxBasis;   /* maximum size of GD */
   int     maxIter   = jd->maxIter;    /* number of maximum iterations of GD */
   int     basisSize = 1;              /* basis size in blocks */
   double  tol       = jd->tol;        /* tolerance of convergence */
   double  normA     = jd->normMatrix; /* norm of sparse matrix */
   //double  maxerr;
   //int     iter      = 0;

   jd->numMatVecsfp64 = 0;
   jd->numMatVecsfp16 = 0;

   double *normr = (double*)malloc(sizeof(double)*numEvals);

   // Step 0.1: Initialize matrices and basis
   initBasis(W,ldW,H,ldH,V,ldV,L, AW, ldAW, dim,maxBasis,numEvals,jd); // basis initilization and H creation

   // Step 0.2: First approximation of eigenpairs
   eigH(V, ldV, L, W,ldW, H, ldH, numEvals, basisSize,jd);  // first approximation of eigevectors
   // Step 0.3: Residual calculation
   residual(R, ldR, V, ldV, L, numEvals, jd); 
   
   /* main loop of JDQMR */
   for(int i=0;i<maxIter;i++){   
      /* Inner sQMR16 to be used here in the future */
      //hipMemcpy(P,R,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
      innerSolver(P,ldP,R,ldR,V,ldV,L,numEvals,dim,jd);

      if(basisSize == maxBasis){
         /* no space left - Restart basis */
         restart(W, ldW, H, ldH, Vprev, ldVprev, NULL ,
                  V, ldV, L, AW, ldAW, &basisSize, maxBasis, numEvals, dim, jd);
      }

      /* Enrich basis with new vectors*/    
      expandBasis(W, ldW, H, ldH, P, ldP, AW, ldAW, basisSize, dim,numEvals, jd);
      //basisSize++;
      
      /* keep previous ritz vectors for restarting purposes*/
      hipMemcpy(Vprev,V,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
      /* Find new Ritz pairs */
      eigH(V, ldV, L, W,ldW, H, ldH, numEvals, basisSize,jd);  // first approximation of eigevectors

      /* Residual calculation */
      residual(R, ldR, V, ldV, L, numEvals, jd); 

      /* convergence check */
      int    numConverged = 0;
      for(int j=0;j<numEvals;j++){
         hipblasDnrm2(jd->gpuH->cublasH,dim,&R[0+j*ldR], 1, &normr[j]);
         if(normr[j] < tol*normA){
            numConverged++;
         }
      }

      #if 1
      for(int i=0;i<numEvals;i++){
         printf("||R[:,%d]||: %e\n",i,normr[i]/normA);
      }
      #endif         

      if(numConverged == numEvals){
         break;
      }
         

      jd->outerIterations++;
         
   }


   /* Get eigenpairs back */

#if 0
for(int i=0;i<numEvals;i++){
   printf("||R[:,%d]||: %e\n",i,normr[i]);
}
printMatrixDouble(L,numEvals,1,"L");

printf("Iterations=%d \nTolerance=%e\nnormA=%e\n",iter,tol,normA);
printf("fp64 matVecs=%d\nfp16 matVecs=%d\n",jd->numMatVecsfp64,jd->numMatVecsfp16);
#endif         


}



void jdqmr16_eigenpairs(double *V, int ldV, double *L, struct jdqmr16Info *jd){
   
   struct devSolverSpace* sp = jd->sp;

   hipMemcpy(V,sp->V,sizeof(double)*(jd->numEvals)*ldV,hipMemcpyDeviceToHost);
   hipMemcpy(L,sp->L,sizeof(double)*(jd->numEvals),hipMemcpyDeviceToHost);


}






























