#include "hip/hip_runtime.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "initBasis.h"
#include "eigH.h"
#include "residual.h"   
#include "expandBasis.h"
#include "restart.h"

void init_jdqmr16(struct jdqmr16Info *jd){
   
   /* allocate gpu memory */
   struct jdqmr16Matrix *A = jd->matrix;   
   
   double *vals    = A->values;
   int    *rows    = A->rows;
   int    *cols    = A->cols;

   double *devVals  = A->devValuesD;
   int    *devRows  = A->devRows;
   int    *devCols  = A->devCols;
   int     nnz      = A->nnz;
   int     dim      = A->dim;
   int     numEvals = jd->numEvals;
   int     maxBasis = jd->maxBasis;

  
   /* if matrix is small */
   if(numEvals*maxBasis >= dim){
      jd->maxBasis = floor(dim/numEvals);
      maxBasis = jd->maxBasis;
   }
 
   /* initialize data to device */
   CUDA_CALL(hipMalloc((void**)&(A->devValuesD),sizeof(double)*nnz));
   CUDA_CALL(hipMalloc((void**)&(A->devCols),sizeof(int)*nnz));
   CUDA_CALL(hipMalloc((void**)&(A->devRows),sizeof(int)*nnz));

   CUDA_CALL(hipMemcpy((void*)(A->devValuesD),(void*)vals,sizeof(double)*nnz,hipMemcpyHostToDevice));
   CUDA_CALL(hipMemcpy((void*)(A->devCols),(void*)cols,sizeof(int)*nnz,hipMemcpyHostToDevice));
   CUDA_CALL(hipMemcpy((void*)(A->devRows),(void*)rows,sizeof(int)*nnz,hipMemcpyHostToDevice));

   /* allocate device memory for solver */
   jd->sp   = (struct devSolverSpace*)malloc(sizeof(struct devSolverSpace));
   jd->gpuH = (struct gpuHandler*)malloc(sizeof(struct gpuHandler));
   
   /* initialize gpu handlers */
   struct gpuHandler *gpuH = jd->gpuH;   
   // hiprand
   hiprandGenerator_t *curandH = &(gpuH->curandH);
	hiprandCreateGenerator(curandH, HIPRAND_RNG_PSEUDO_DEFAULT);
   // cusolver
   hipsolverHandle_t *cusolverH = &(gpuH->cusolverH);
   hipsolverDnCreate(cusolverH);
   // cublas
   hipblasHandle_t *cublasH =  &(gpuH->cublasH);
   hipblasCreate(cublasH);   
   // cusparse
   hipsparseHandle_t *cusparseH = &(gpuH->cusparseH);
   hipsparseCreate(cusparseH);

   /* initialize space for solver */
   struct devSolverSpace* sp = jd->sp;
   CUDA_CALL(hipMalloc((void**)&sp->W,sizeof(double)*dim*maxBasis*numEvals));               sp->ldW     = dim;
   CUDA_CALL(hipMalloc((void**)&sp->H,sizeof(double)*maxBasis*numEvals*maxBasis*numEvals)); sp->ldH     = maxBasis*numEvals;
   CUDA_CALL(hipMalloc((void**)&sp->Vprev,sizeof(double)*numEvals*dim));                    sp->ldVprev = dim;
   CUDA_CALL(hipMalloc((void**)&sp->V,sizeof(double)*numEvals*dim));                        sp->ldV     = dim;
   CUDA_CALL(hipMalloc((void**)&sp->Lprev,sizeof(double)*numEvals)); 
   CUDA_CALL(hipMalloc((void**)&sp->L,sizeof(double)*numEvals)); 
   CUDA_CALL(hipMalloc((void**)&sp->R,sizeof(double)*numEvals*dim));                        sp->ldR     = dim;

   CUDA_CALL(hipMalloc((void**)&sp->AW,sizeof(double)*maxBasis*numEvals*dim));              sp->ldAW    = dim;
   //CUDA_CALL(hipMalloc((void**)&sp->AP,sizeof(double)*numEvals*dim));                       sp->ldAP    = dim;
   CUDA_CALL(hipMalloc((void**)&sp->P,sizeof(double)*numEvals*dim));                        sp->ldP     = dim;
   

   double *H        = sp->H;        /* projected Matrix */
   double *V        = sp->V;        /* Ritz vectors */
   double *W        = sp->L;        /* Ritz values */
   
   /* init inner functions */

   // init initBasis
   jd->spInitBasis = (struct initBasisSpace *)malloc(sizeof(struct initBasisSpace));
   initBasis_init(sp->W,sp->ldW, sp->H, sp->ldH, sp->V, sp->ldV,sp->L, dim, maxBasis,numEvals,jd);

   // init eigH
   jd->spEigH = (struct eigHSpace *)malloc(sizeof(struct eigHSpace));   
   eigH_init(sp->W, sp->ldW, sp->L, sp->H, sp->ldH, numEvals, maxBasis, jd);
   
   // init residual
   jd->spResidual = (struct residualSpace *)malloc(sizeof(struct residualSpace));
   residual_init(sp->R,sp->ldR,sp->V,sp->ldV,sp->L,numEvals,jd);

   // init expandBasis
   jd->spExpandBasis = (struct expandBasisSpace *)malloc(sizeof(struct expandBasisSpace));
   expandBasis_init(sp->W, sp->ldW, sp->H, sp->ldH, sp->P, sp->ldP,maxBasis, dim, numEvals, jd);

   // init restart
   jd->spRestart = (struct restartSpace *)malloc(sizeof(struct restartSpace));
   restart_init(sp->W, sp->ldW, sp->H, sp->ldH, sp->Vprev, sp->ldVprev, sp->Lprev, sp->V, sp->ldV, sp->L,
               &maxBasis, maxBasis, numEvals, dim, jd);

   return;
}

void destroy_jdqmr16(struct jdqmr16Info *jd){

   /* destroy inner functions */
   restart_destroy(jd);
   expandBasis_destroy(jd);
   initBasis_destroy(jd);
   eigH_destroy(jd);
   residual_destroy(jd);

   /* destroy gpu handlers */ 
   struct gpuHandler *gpuH = jd->gpuH;   

   // hiprand     
   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandDestroyGenerator(curandH);
   // cusolver
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
	hipsolverDnDestroy(cusolverH);
	// cublas
   hipblasHandle_t cublasH = gpuH->cublasH;
   hipblasDestroy(cublasH);
   // cusparse
   hipsparseHandle_t cusparseH = gpuH->cusparseH;
   hipsparseDestroy(cusparseH);

   /* Destroy Matrix */
   struct jdqmr16Matrix *A = jd->matrix;   
   
   double *devVals = A->devValuesD;
   int    *devRows = A->devRows;
   int    *devCols = A->devCols;
   int    nnz      = A->nnz;
   int    dim      = A->dim;

   CUDA_CALL(hipFree(devVals));
   CUDA_CALL(hipFree(devCols));
   CUDA_CALL(hipFree(devRows));


   struct devSolverSpace *sp = jd->sp;


   CUDA_CALL(hipFree(sp->AW));
   CUDA_CALL(hipFree(sp->P));
   //CUDA_CALL(hipFree(sp->AP));
   CUDA_CALL(hipFree(sp->R));
   CUDA_CALL(hipFree(sp->W));
   CUDA_CALL(hipFree(sp->H));
   CUDA_CALL(hipFree(sp->Vprev));
   CUDA_CALL(hipFree(sp->V));
   CUDA_CALL(hipFree(sp->L));
   CUDA_CALL(hipFree(sp->Lprev));

   free(jd->spRestart);
   free(jd->spExpandBasis);
   free(jd->spInitBasis);
   free(jd->spEigH);
   free(jd->spResidual);

   free(jd->gpuH);
   free(jd->sp);
   
   return;

}

void jdqmr16(struct jdqmr16Info *jd){

   /* Generalized Davidson Iteration */
   struct jdqmr16Matrix *A = jd->matrix;   

   struct devSolverSpace* sp = jd->sp;
   double *W  = sp->W;  int ldW = sp->ldW; /* GD basis */
   double *H  = sp->H;  int ldH = sp->ldH; /* projected Matrix */
   double *V  = sp->V;  int ldV = sp->ldV; /* Ritz vectors */
   double *Vprev  = sp->Vprev;  int ldVprev = sp->ldVprev; /* Ritz vectors */
   
   double *L      = sp->L;                     /* Ritz values */
   double *Lprev  = sp->Lprev;                 /* Ritz values */
   
   double *R  = sp->R;  int ldR = sp->ldR; /* Ritz vectors */
   double *P  = sp->P;  int ldP = sp->ldP;
   double *AW = sp->AW; int ldAW = sp->ldAW;

   int     dim      = A->dim;       /* dimension of the problem */
   int     numEvals = jd->numEvals; /* number of wanted eigenvalues */
   int     maxBasis = jd->maxBasis; /* maximum size of GD */
   int     maxIter  = jd->maxIter;  /* number of maximum iterations of GD */
   int     basisSize = 1;           /* basis size in blocks */

   // Step 0.1: Initialize matrices and basis
   initBasis(W,ldW,H,ldH,V,ldV,L, AW, ldAW, dim,maxBasis,numEvals,jd); // basis initilization and H creation

   // Step 0.2: First approximation of eigenpairs
   eigH(V, ldV, L, W,ldW, H, ldH, numEvals, basisSize,jd);  // first approximation of eigevectors
   // Step 0.3: Residual calculation
   residual(R, ldR, V, ldV, L, numEvals, jd); 
   

   /* main loop of JDQMR */
   for(int i=0;i<maxIter;i++){   

      /* Inner sQMR16 to be used here in the future */
      hipMemcpy(P,R,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);

      if(basisSize == maxBasis){
  
//         printMatrixDouble(Vprev,ldVprev,numEvals,"Vprev");
//         printMatrixDouble(V,ldV,numEvals,"V");
         restart(W, ldW, H, ldH, Vprev, ldVprev, Lprev,
                  V, ldV, L, &basisSize, maxBasis, numEvals, dim, jd);
//         printMatrixDouble(W,ldW,basisSize*numEvals,"W");   
//         printMatrixDouble(H,ldH,ldH,"H");
         //return;
//         break;
         printMatrixDouble(L,numEvals,1,"L");
//         printf("Restart\n");
      }

      /* Enrich basis with new vectors*/    
      expandBasis(W, ldW, H, ldH, P, ldP, AW, ldAW, basisSize, dim,numEvals, jd);
      basisSize++;

      /* keep previous ritz vectors for restarting purposes*/
      hipMemcpy(Vprev,V,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
      hipMemcpy(Lprev,L,sizeof(double)*numEvals,hipMemcpyDeviceToDevice);
      /* Find new Ritz pairs */
      eigH(V, ldV, L, W,ldW, H, ldH, numEvals, basisSize,jd);  // first approximation of eigevectors
      
      /* Residual calculation */
      residual(R, ldR, V, ldV, L, numEvals, jd); 

      
   }



#if 1
//if(i>3){
//   printMatrixDouble(V,ldV,numEvals,"V");
   printMatrixDouble(L,numEvals,1,"L");
 //  printMatrixDouble(Vprev,ldVprev,numEvals,"Vprev");
//   printMatrixDouble(Lprev,numEvals,1,"Lprev");
   //printMatrixDouble(R,ldR,numEvals,"R");
//   printMatrixDouble(W,ldW,basisSize*numEvals,"W");
//   printMatrixDouble(H,ldH,ldH,"H");

//}
#endif
         


}
































