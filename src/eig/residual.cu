#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdio.h>
#include <assert.h>

#include "residual.h"

#include "../include/helper.h"
#include "../../include/jdqmr16.h"



void residual_init(double *R, int ldR, double *V, int ldV, double *L, int numEvals, struct jdqmr16Info *jd){
 
   struct jdqmr16Matrix  *A = jd->matrix;
   int dim    = A->dim;

   /* handlers for gpu and jdqmr16 */
   struct gpuHandler *gpuH    = jd->gpuH;
   struct eigHSpace  *spEig   = jd->spEigH;
   struct residualSpace *spRes = jd->spResidual;
   
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;
   hipsparseHandle_t   cusparseH = gpuH->cusparseH;



   hipMalloc((void**)&(spRes->VL),sizeof(double)*dim*numEvals);
   spRes->ldVL = dim;
   spRes->hL = (double*)malloc(sizeof(double)*numEvals);


   // R = AV
   hipsparseCreateCoo(&(spRes->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
	hipsparseCreateDnMat(&(spRes->descrV),dim,numEvals,ldV,(void*)V,HIP_R_64F,HIPSPARSE_ORDER_COL);
	hipsparseCreateDnMat(&(spRes->descrR),dim,numEvals,ldR,(void*)R,HIP_R_64F,HIPSPARSE_ORDER_COL);

   double one  = 1.0;
   double zero = 0.0;

   hipsparseSpMM_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spRes->descrA,spRes->descrV,&zero,
                        spRes->descrR,HIP_R_64F,HIPSPARSE_COOMM_ALG2,&(spRes->bufferSize));


   assert(spRes->bufferSize>0);
	hipMalloc((void**)&(spRes->buffer),spRes->bufferSize);

}

void residual_destroy(struct jdqmr16Info *jd){

   struct residualSpace *spRes = jd->spResidual;
   
   hipFree(spRes->buffer);
   hipFree(spRes->VL);
   free(spRes->hL);


}


void residual(double *R, int ldR, double *V, int ldV, double *L, int numEvals, struct jdqmr16Info *jd){


   
   struct jdqmr16Matrix  *A = jd->matrix;
   int dim    = A->dim;

   /* handlers for gpu and jdqmr16 */
   struct gpuHandler *gpuH    = jd->gpuH;
   struct eigHSpace  *spEig   = jd->spEigH;
   struct residualSpace *spRes = jd->spResidual;
   
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   hipblasHandle_t     cublasH   = gpuH->cublasH;
   hipsparseHandle_t   cusparseH = gpuH->cusparseH;
   double *VL = spRes->VL; int ldVL = spRes->ldVL;
   
   
   // buffer matrices allocation mem
   // AV,VL, hL
   double *hL = spRes->hL;
   hipMemcpy(spRes->hL,L,sizeof(double)*numEvals,hipMemcpyDeviceToHost); 

   // VL = V
   hipMemcpy(spRes->VL,V,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice); 
   // VL = VL*L
   double *alpha,*x;
	int incx = 1;

   hipblasDdgmm(cublasH,HIPBLAS_SIDE_RIGHT,dim,numEvals,VL,ldVL,L,1,VL,ldVL);
/*
   for(int i=0;i<numEvals;i++){
		
		alpha = &hL[i];
		x 	   = &(spRes->VL[0+spRes->ldVL*i]);
		
		hipblasDscal(cublasH,dim,alpha,x,incx);
	}
*/
   // R = AV

   double one  = 1.0;
   double zero = 0.0;

   hipsparseSpMM(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,spRes->descrA,spRes->descrV,&zero,spRes->descrR,HIP_R_64F,HIPSPARSE_COOMM_ALG2,spRes->buffer);


   /* R = R-VL */
   double minus_one = -1.0;
   hipblasDgeam(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N,dim,numEvals,&one,R, ldR,
                          &minus_one,spRes->VL,spRes->ldVL,R,ldR);

}
