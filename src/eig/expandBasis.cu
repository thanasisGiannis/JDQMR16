#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdio.h>
#include <assert.h>

#include "expandBasis.h"

#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void expandBasis_init(double *W, int ldW, double *H, int ldH, double *P, int ldP,
                int maxBasisSize, int dim, int numEvals, struct jdqmr16Info *jd){

   struct expandBasisSpace *spExpandBasis = jd->spExpandBasis;


   hipMalloc((void**)&(spExpandBasis->AP),sizeof(double)*dim*numEvals);
   spExpandBasis->ldAP = dim;
   
   /* P = P - W*W'*P */
   struct gpuHandler     *gpuH      = jd->gpuH;
   hipblasHandle_t         cublasH   = gpuH->cublasH;
   hipsolverHandle_t     cusolverH = gpuH->cusolverH;
   hipsparseHandle_t       cusparseH = gpuH->cusparseH;


   hipMalloc((void**)&(spExpandBasis->WTP),sizeof(double)*maxBasisSize*numEvals*numEvals);
   spExpandBasis->ldWTP = maxBasisSize*numEvals;

   /* P = orth(P) */
   double *d_tau   = spExpandBasis->d_tau;
   int    *devInfo = spExpandBasis->devInfo;
   double *d_work  = spExpandBasis->d_work;

   int lwork_geqrf = 0;
   int lwork_orgqr = 0;
   int lwork = spExpandBasis->lwork;
   int info_gpu = 0;   

   hipMalloc((void**)&(spExpandBasis->d_tau), sizeof(double)*dim);
   hipMalloc((void**)&(spExpandBasis->devInfo), sizeof(int));

   
   hipsolverDnDgeqrf_bufferSize(cusolverH,dim,numEvals,P,ldP,&lwork_geqrf);
   hipsolverDnDorgqr_bufferSize(cusolverH,dim,numEvals,numEvals,P,ldP,spExpandBasis->d_tau,&lwork_orgqr);

   spExpandBasis->lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
   hipMalloc((void**)&(spExpandBasis->d_work), sizeof(double)*(spExpandBasis->lwork));



   /* AP = A*P */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spExpandBasis->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);
	hipsparseCreateDnMat(&(spExpandBasis->descrP),dim,numEvals,ldP,(void*)P,HIP_R_64F,HIPSPARSE_ORDER_COL);
	hipsparseCreateDnMat(&(spExpandBasis->descrAP),dim,numEvals,spExpandBasis->ldAP,(void*)spExpandBasis->AP,HIP_R_64F,HIPSPARSE_ORDER_COL);

   double one  = 1.0;
   double zero = 0.0;

   hipsparseSpMM_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spExpandBasis->descrA,spExpandBasis->descrP,&zero,
                        spExpandBasis->descrAP,HIP_R_64F,HIPSPARSE_COOMM_ALG2,&(spExpandBasis->bufferSize));


   assert(spExpandBasis->bufferSize>0);
	hipMalloc((void**)&(spExpandBasis->buffer),spExpandBasis->bufferSize);


}

void expandBasis_destroy(struct jdqmr16Info *jd){

   struct expandBasisSpace  *spExpandBasis = jd->spExpandBasis;

   hipFree(spExpandBasis->WTP);
   hipFree(spExpandBasis->d_tau);
   hipFree(spExpandBasis->devInfo);
   hipFree(spExpandBasis->d_work);

   
   hipFree(spExpandBasis->buffer);
   
   hipFree(spExpandBasis->AP);

}

void expandBasis(double *W, int ldW, double *H, int ldH, double *P, int ldP, double *Qlocked, int ldQlocked, int numLocked,
                double *AW, int ldAW, int &basisSize, int dim, int numEvals, struct jdqmr16Info *jd){

   
   struct gpuHandler        *gpuH          = jd->gpuH;
   struct expandBasisSpace  *spExpandBasis = jd->spExpandBasis;

   hipblasHandle_t         cublasH   = gpuH->cublasH;
   hipsolverHandle_t     cusolverH = gpuH->cusolverH;
   hipsparseHandle_t       cusparseH = gpuH->cusparseH;

   /* P = -W*W'*P + P */
   double *WTP = spExpandBasis->WTP; int ldWTP = spExpandBasis->ldWTP;
   
   double one  = 1.0;
   double zero = 0.0;

//   for(int i=0; i<basisSize; i++){
   for(int i=0; i<1; i++){
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,basisSize*numEvals,numEvals,dim,&one,
                              W,HIP_R_64F,ldW,P,HIP_R_64F,ldP,&zero,
                              WTP,HIP_R_64F,ldWTP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      double minus_one = -1.0;
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,basisSize*numEvals,&minus_one,
                              W,HIP_R_64F,ldW,WTP,HIP_R_64F,ldWTP,&one,
                              P,HIP_R_64F,ldP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
   }


   /* P = - Qlocked*Qlocked'*P + P  */
   for(int i=0; i<1; i++){
      // using W'P same buffer space
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numLocked,numEvals,dim,&one,
                              Qlocked,HIP_R_64F,ldQlocked,P,HIP_R_64F,ldP,&zero,
                              WTP,HIP_R_64F,ldWTP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
      double minus_one = -1.0;
      CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,numEvals,numLocked,&minus_one,
                              Qlocked,HIP_R_64F,ldQlocked,WTP,HIP_R_64F,ldWTP,&one,
                              P,HIP_R_64F,ldP,HIP_R_64F,
                              HIPBLAS_GEMM_DEFAULT));
   }

   /* P = orth(P)  */
   double *d_tau   = spExpandBasis->d_tau;
   int    *devInfo = spExpandBasis->devInfo;
   double *d_work  = spExpandBasis->d_work;
   int     lwork   = spExpandBasis->lwork;

   hipsolverDnDgeqrf(cusolverH,dim,numEvals,P,ldP,d_tau,d_work,lwork,devInfo);
   hipsolverDnDorgqr(cusolverH,dim,numEvals,numEvals,P,ldP,d_tau,d_work,lwork,devInfo);

   /* AP = A*P */
   double *AP = spExpandBasis->AP; int ldAP = spExpandBasis->ldAP;
   hipsparseSpMM(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,spExpandBasis->descrA,spExpandBasis->descrP,&zero,spExpandBasis->descrAP,HIP_R_64F,
             HIPSPARSE_COOMM_ALG2,spExpandBasis->buffer);

   /* H = [H W'*AP; P'*AW P'*AP*/
   // P'*AP   
   CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals,dim,&one,
                           P,HIP_R_64F,ldP,AP,HIP_R_64F,ldAP,&zero,
                           &H[basisSize*numEvals+(basisSize*numEvals)*ldH],HIP_R_64F,ldH,HIP_R_64F,
                           HIPBLAS_GEMM_DEFAULT));


   // W'*AP
   CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,basisSize*numEvals,numEvals,dim,&one,
                           W,HIP_R_64F,ldW,AP,HIP_R_64F,ldAP,&zero,
                           &H[0+(basisSize*numEvals)*ldH],HIP_R_64F,ldH,HIP_R_64F,
                           HIPBLAS_GEMM_DEFAULT));


   // P'*AW
   CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,basisSize*numEvals,dim,&one,
                           P,HIP_R_64F,ldP,AW,HIP_R_64F,ldAW,&zero,
                           &H[basisSize*numEvals+0*ldH],HIP_R_64F,ldH,HIP_R_64F,
                           HIPBLAS_GEMM_DEFAULT));


   /* AW = [AW AP] */
   hipMemcpy(&AW[0 + basisSize*numEvals*ldAW],AP,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);
   /* W = [W P] */
   hipMemcpy(&W[0 +basisSize*numEvals*ldW],P,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);


   basisSize++;
}














