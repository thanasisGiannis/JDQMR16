#include "hip/hip_runtime.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"


#include <hiprand.h>
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "sqmr.h"

void sqmr_init(half *X, int ldX, half *B, int ldB, int dim, double infNormB, struct jdqmr16Info *jd){

 
   half tmpScalar;
   half minus_one = __float2half(-1.0);
   half one  = __float2half(1.0);
   half zero = __float2half(0.0);



   half *x = X;
   half *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   half *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(half)*dim);

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesH,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_16F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)spsqmr->d,HIP_R_16F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)spsqmr->w,HIP_R_16F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_16F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);



   assert(spsqmr->bufferSize>=0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);


}

void sqmr_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);

}

void sqmr(half *X, int ldX, half *B, int ldB, int dim, double infNormB, struct jdqmr16Info *jd){

/*
   Solving AX=B with sQMR and early stopping criteria

*/

   // this should be input in this function
   float ita    = 0.0;
   float thita_ = 0.0; 
   int qmrMaxIt = 1000;
   int tol      = 1e-08;

   
   float Thita_ = 0.0;
   float rho_;
   float sigma;
   float alpha;  
   float normr;
   float Thita;
   float c;
   float g_;
   float r00;
   float rho;
   float vita;
   float g;
   float gama;
   float xi;
   float normt; 
   float f;
   float p;
   float thita;
   float pk;
   float rkm;
   float scalw;
      

   float minus_alpha;
   float deltaScal1;
   float deltaScal2;
   float one32 = 1.0;
      
   float BITA  = 0.0;
   float DELTA = 0.0;
   float GAMA  = 0.0;
   float FI    = 0.0;
   float PSI   = 0.0;


   half tmpScalar;
   half minus_one = __float2half(-1.0);
   half one  = __float2half(1.0);
   half zero = __float2half(0.0);



   half *x = X;
   half *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;

   hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);
   hipDeviceSynchronize();

#if 0
   half *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(half)*dim);

   half *delta = (half*)spsqmr->delta;
   half *r = (half*)spsqmr->r;     
   half *d = (half*)spsqmr->d;     
   half *w = (half*)spsqmr->w;     

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesH,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_16F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)d,HIP_R_16F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)w,HIP_R_16F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_16F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);


   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;

   assert(spsqmr->bufferSize>=0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);

#else


   half *t = x;
   half *delta = (half*)spsqmr->delta;
   half *r = (half*)spsqmr->r;     
   half *d = (half*)spsqmr->d;     
   half *w = (half*)spsqmr->w;     

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;


#endif

   /* r = -b */
   hipMemcpy(r,b,sizeof(half)*dim,hipMemcpyDeviceToDevice);
   hipblasScalEx(cublasH,dim,&minus_one,HIP_R_16F,r,HIP_R_16F,1,HIP_R_32F);

   /* d = r */
   hipMemcpy(d,r,sizeof(half)*dim,hipMemcpyDeviceToDevice);
   hipblasNrm2Ex(cublasH,dim,b,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F);
   g = __half2float(tmpScalar);


   hipblasDotEx(cublasH,dim,(void*)r,HIP_R_16F,1,(void*)d,HIP_R_16F,1,(void*)&tmpScalar,HIP_R_16F,HIP_R_32F);
   rho_ = __half2float(tmpScalar);

   /* sQMR main iteration */
   for(int i=0; i<qmrMaxIt; i++){
      
      /* w = A*d */
      hipsparseSpMV(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrd,&zero,descrw,HIP_R_16F,
             HIPSPARSE_COOMV_ALG,buffer);


      /* sigma = d'*w */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,d,HIP_R_16F,1,w,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
      sigma = __half2float(tmpScalar);
      /* alpha = rho_/sigma */
      alpha =1*rho_/sigma;
   

      /* r = r -alpha*w */
      minus_alpha = -1*alpha;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_alpha,HIP_R_32F,w,HIP_R_16F,1,r,HIP_R_16F,1,HIP_R_32F));

      /* Thita = norm(r)/g */
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
      normr = __half2float(tmpScalar);
      Thita = normr/g;

      /* c = 1./sqrt(1+Thita*Thita) */
      c = sqrt(1/(1+Thita*Thita));
      /* g = g*Thita*c */
      g =g*Thita*c;

      if(i == 0){
         g_ = g;
      }

      /* delta = (c*c*alpha)*d + (c*c*Thita_*Thita_)*delta  */
      deltaScal1 = c*c*Thita_*Thita_;   
      deltaScal2 = c*c*alpha;
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&deltaScal1,HIP_R_32F,delta,HIP_R_16F,1,HIP_R_32F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&deltaScal2,HIP_R_32F,d,HIP_R_16F,1,delta,HIP_R_16F,1,HIP_R_32F));
      /*  t  = t + delta */
      
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one32,HIP_R_32F,delta,HIP_R_16F,1,t,HIP_R_16F,1,HIP_R_32F));

      if(abs(g) < tol){
         break;
      }

      gama = c*c*Thita_; 
      xi = c*c*alpha;    
      normt; 
      hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F); 
      normt = __half2float(tmpScalar);
      f = 1 + normt*normt; 
      PSI = gama*(PSI + FI);
      FI = gama*gama*FI + xi*xi*sigma;
      GAMA = GAMA  + 2*PSI + FI;


      DELTA = gama*DELTA - xi*rho_;
      BITA = BITA + DELTA;
      p = ((thita_-ita+2*BITA+GAMA))/f;
      thita = ita+p;

      pk = (((thita_)-ita+BITA)*((thita_)-ita+BITA))/f - p*p;
      rkm = sqrt(((g)*(g))/f + pk);


      if(i==0){
         r00 = rkm;
      }


      if(rho_ < tol){
         break;
      }


      rkm = sqrt(g*g/f);

       if( (g < rkm*max(0.99 * sqrt(f),sqrt(g/g_))) || (thita > thita_) || rkm<0.1*r00  || g < tol || rkm < tol){
          break; 
       }

      /*  w = r./norm(r); */
      hipMemcpy(w,r,sizeof(half)*dim,hipMemcpyDeviceToDevice);
      hipblasNrm2Ex(cublasH,dim,w,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F); 
      scalw = 1/__half2float(tmpScalar);
      hipblasScalEx(cublasH,dim,&scalw,HIP_R_32F,w,HIP_R_16F,1,HIP_R_32F);

         
      /* rho = r'*w; */
      hipblasDotEx(cublasH,dim,(void*)r,HIP_R_16F,1,(void*)w,HIP_R_16F,1,(void*)&tmpScalar,HIP_R_16F,HIP_R_32F);
      rho = __half2float(tmpScalar);
      vita = rho/rho_;



      /* d = w + vita*d; */
      hipblasScalEx(cublasH,dim,&vita,HIP_R_32F,d,HIP_R_16F,1,HIP_R_32F);
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one32,HIP_R_32F,w,HIP_R_16F,1,d,HIP_R_16F,1,HIP_R_32F));


      thita_ = thita;
      Thita_ = Thita;
      rho_ = rho;
      g_ = g;

   }

#if 0
   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);
#endif
}
