#include "hip/hip_runtime.h"
#include "../../include/jdqmr16.h"
#include "../include/helper.h"


#include <hiprand.h>
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "sqmr.h"


void sqmrF(float *X, int ldX, float *B, int ldB, int dim, double infNormB, struct jdqmr16Info *jd){
/*
   Solving AX=B with sQMR and early stopping criteria

*/
   // this should be input in this function
   float ita    = 0.0;
   float thita_ = 0.0; 
   int qmrMaxIt  = 100000000;// max(10*dim,1000);
   float tol    = 1e-08;

   
   float Thita_ = 0.0;
   float rho_;
   float sigma;
   float alpha;  
   float normr;
   float Thita;
   float c;
   float g_;
   float r00;
   float rho;
   float vita;
   float g;
   float gama;
   float xi;
   float normt; 
   float f;
   float p;
   float thita;
   float pk;
   float rkm;
   float scalw;
      

   float minus_alpha;
   float deltaScal1;
   float deltaScal2;
   float one32 = 1.0;
      
   float BITA  = 0.0;
   float DELTA = 0.0;
   float GAMA  = 0.0;
   float FI    = 0.0;
   float PSI   = 0.0;


   float minus_one = -1.0;
   float one       =  1.0;
   float zero      =  0.0;



   float *x = X;
   float *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;

   hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);
   hipDeviceSynchronize();


   float *t = x;
   hipMemset(t,0,sizeof(float)*dim);
   float *delta = (float*)spsqmr->delta;
   float *r     = (float*)spsqmr->r;     
   float *d     = (float*)spsqmr->d;     
   float *w     = (float*)spsqmr->w;     
   float *VTd   = (float*)spsqmr->VTd;

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;


   /* r = 0*r -b */
   hipMemset(r,0,sizeof(float)*dim);
   CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_one,HIP_R_32F,b,HIP_R_32F,1,r,HIP_R_32F,1,HIP_R_32F));

   /* d = r */
   hipMemset(d,0,sizeof(float)*dim);
   CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_32F,r,HIP_R_32F,1,d,HIP_R_32F,1,HIP_R_32F));


   CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,b,HIP_R_32F,1,&g,HIP_R_32F,HIP_R_32F));


   CUBLAS_CALL(hipblasDotEx(cublasH,dim,(void*)r,HIP_R_32F,1,(void*)d,HIP_R_32F,1,(void*)&rho_,HIP_R_32F,HIP_R_32F));
   /* sQMR main iteration */
   for(int i=0; i<qmrMaxIt; i++){
      jd->innerIterations++;
      /* w = A*d */
      hipsparseSpMV(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrd,&zero,descrw,HIP_R_32F,
             HIPSPARSE_COOMV_ALG,buffer);
      jd->numMatVecsfp16++;

      /* sigma = d'*w */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,d,HIP_R_32F,1,w,HIP_R_32F,1,&sigma,HIP_R_32F,HIP_R_32F));
      /* alpha = rho_/sigma */
      alpha =1*rho_/sigma;
   

      /* r = r -alpha*w */
      minus_alpha = -1*alpha;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_alpha,HIP_R_32F,w,HIP_R_32F,1,r,HIP_R_32F,1,HIP_R_32F));

      /* Thita = norm(r)/g */
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_32F,1,&normr,HIP_R_32F,HIP_R_32F));
      Thita = normr/g;

      /* c = 1./sqrt(1+Thita*Thita) */
      c = sqrt(1/(1+Thita*Thita));
      /* g = g*Thita*c */
      g =g*Thita*c;

      if(i == 0){
         g_ = g;
      }

      /* delta = (c*c*alpha)*d + (c*c*Thita_*Thita_)*delta  */
      deltaScal1 = c*c*Thita_*Thita_;   
      deltaScal2 = c*c*alpha;
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&deltaScal1,HIP_R_32F,delta,HIP_R_32F,1,HIP_R_32F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&deltaScal2,HIP_R_32F,d,HIP_R_32F,1,delta,HIP_R_32F,1,HIP_R_32F));
      /*  t  = t + delta */
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_one,HIP_R_32F,delta,HIP_R_32F,1,t,HIP_R_32F,1,HIP_R_32F));

      if(abs(g) < tol ){
         break;
      }

      gama = c*c*Thita_; 
      xi = c*c*alpha;    
      normt; 
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_32F,1,&normt,HIP_R_32F,HIP_R_32F)); 
      f = 1 + normt*normt; 
      PSI = gama*(PSI + FI);
      FI = gama*gama*FI + xi*xi*sigma;
      GAMA = GAMA  + 2*PSI + FI;


      DELTA = gama*DELTA - xi*rho_;
      BITA = BITA + DELTA;
      p = ((thita_-ita+2*BITA+GAMA))/f;
      thita = ita+p;

      pk = (((thita_)-ita+BITA)*((thita_)-ita+BITA))/f - p*p;
      rkm = sqrt(((g)*(g))/f + pk);


      if(i==0){
         r00 = rkm;
      }


      if(rho_ < tol ){
         break;
      }


      rkm = sqrt(g*g/f);

       if(((g < rkm*max(0.99 * sqrt(f),sqrt(g/g_))) || (thita > thita_) || rkm<0.1*r00  || g < tol || rkm < tol)){
         break; 
       }

      hipMemset(w,0,sizeof(float)*dim);
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_32F,1,&scalw,HIP_R_32F,HIP_R_32F)); 
      scalw = 1/scalw;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&scalw,HIP_R_32F,r,HIP_R_32F,1,w,HIP_R_32F,1,HIP_R_32F));

         
      /* rho = r'*w; */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,(void*)r,HIP_R_32F,1,(void*)w,HIP_R_32F,1,(void*)&rho,HIP_R_32F,HIP_R_32F));
      vita = rho/rho_;

      /* d = w + vita*d; */
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&vita,HIP_R_32F,d,HIP_R_32F,1,HIP_R_32F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_32F,w,HIP_R_32F,1,d,HIP_R_32F,1,HIP_R_32F));


      thita_ = thita;
      Thita_ = Thita;
      rho_ = rho;
      g_ = g;

   }

}


void sqmrF_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);

}



void sqmrF_init(float *X, int ldX, float *B, int ldB, int dim, double infNormB, struct jdqmr16Info *jd){

 
   float tmpScalar;
   float minus_one = -1.0;
   float one  = 1.0;
   float zero = 0.0;



   float *x = X;
   float *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   float *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(float)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(float)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(float)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(float)*dim);

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesF,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_32F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)spsqmr->d,HIP_R_32F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)spsqmr->w,HIP_R_32F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_32F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);



   assert(spsqmr->bufferSize>=0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);


}


void sqmrD(double *X, int ldX, double *B, int ldB, double *V,int ldV, int numEvals,
            int dim, double infNormB, struct jdqmr16Info *jd){
/*
   Solving AX=B with sQMR and early stopping criteria

*/

   
   // this should be input in this function
   double ita    = 0.0;
   double thita_ = 0.0; 
   int qmrMaxIt  = max(10*dim,1000);
   double tol    = 1e-08;

   
   double Thita_ = 0.0;
   double rho_;
   double sigma;
   double alpha;  
   double normr;
   double Thita;
   double c;
   double g_;
   double r00;
   double rho;
   double vita;
   double g;
   double gama;
   double xi;
   double normt; 
   double f;
   double p;
   double thita;
   double pk;
   double rkm;
   double scalw;
      

   double minus_alpha;
   double deltaScal1;
   double deltaScal2;
   double one32 = 1.0;
      
   double BITA  = 0.0;
   double DELTA = 0.0;
   double GAMA  = 0.0;
   double FI    = 0.0;
   double PSI   = 0.0;


   double minus_one = -1.0;
   double one       =  1.0;
   double zero      =  0.0;



   double *x = X;
   double *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;

   hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);
   hipDeviceSynchronize();


   double *t = x;
   double *delta = (double*)spsqmr->delta;
   double *r     = (double*)spsqmr->r;     
   double *d     = (double*)spsqmr->d;     
   double *w     = (double*)spsqmr->w;     
   double *VTd   = (double*)spsqmr->VTd;

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;


   /* r = 0*r -b */
   hipMemset(r,0,sizeof(double)*dim);
   CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_one,HIP_R_64F,b,HIP_R_64F,1,r,HIP_R_64F,1,HIP_R_64F));

   /* d = r */
   hipMemset(d,0,sizeof(double)*dim);
   CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_64F,r,HIP_R_64F,1,d,HIP_R_64F,1,HIP_R_64F));


   CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,b,HIP_R_64F,1,&g,HIP_R_64F,HIP_R_64F));


   CUBLAS_CALL(hipblasDotEx(cublasH,dim,(void*)r,HIP_R_64F,1,(void*)d,HIP_R_64F,1,(void*)&rho_,HIP_R_64F,HIP_R_64F));
   int exitLoop = 0;
   /* sQMR main iteration */
   for(int i=0; i<qmrMaxIt; i++){
      jd->innerIterations++;
      /* d = d - VVTd */
      if (V!=0){
         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,1,dim,&one,
                                 V, HIP_R_64F,ldV,d,HIP_R_64F,dim,
                                 &zero,VTd,HIP_R_64F,numEvals,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
         

         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,1,numEvals,&minus_one,
                                 V, HIP_R_64F,ldV,VTd,HIP_R_64F,numEvals,
                                 &one,d,HIP_R_64F,dim,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
      }
      /* w = A*d */
      hipsparseSpMV(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrd,&zero,descrw,HIP_R_64F,
             HIPSPARSE_COOMV_ALG,buffer);
      jd->numMatVecsfp64++;
      /* w = w-VVTw */
      if (V!=0){
         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,1,dim,&one,
                                 V, HIP_R_64F,ldV,w,HIP_R_64F,dim,
                                 &zero,VTd,HIP_R_64F,numEvals,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
         

         CUBLAS_CALL(hipblasGemmEx(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,1,numEvals,&minus_one,
                                 V, HIP_R_64F,ldV,VTd,HIP_R_64F,numEvals,
                                 &one,w,HIP_R_64F,dim,HIP_R_64F,
                                 HIPBLAS_GEMM_DEFAULT));
      }
      /* sigma = d'*w */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,d,HIP_R_64F,1,w,HIP_R_64F,1,&sigma,HIP_R_64F,HIP_R_64F));
      /* alpha = rho_/sigma */
      alpha =1*rho_/sigma;
   

      /* r = r -alpha*w */
      minus_alpha = -1*alpha;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_alpha,HIP_R_64F,w,HIP_R_64F,1,r,HIP_R_64F,1,HIP_R_64F));

      /* Thita = norm(r)/g */
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_64F,1,&normr,HIP_R_64F,HIP_R_64F));
      Thita = normr/g;

      /* c = 1./sqrt(1+Thita*Thita) */
      c = sqrt(1/(1+Thita*Thita));
      /* g = g*Thita*c */
      g =g*Thita*c;

      if(i == 0){
         g_ = g;
      }

      /* delta = (c*c*alpha)*d + (c*c*Thita_*Thita_)*delta  */
      deltaScal1 = c*c*Thita_*Thita_;   
      deltaScal2 = c*c*alpha;
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&deltaScal1,HIP_R_64F,delta,HIP_R_64F,1,HIP_R_64F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&deltaScal2,HIP_R_64F,d,HIP_R_64F,1,delta,HIP_R_64F,1,HIP_R_64F));
      /*  t  = t + delta */
      
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_64F,delta,HIP_R_64F,1,t,HIP_R_64F,1,HIP_R_64F));

      if(abs(g) < tol && exitLoop == 0){
         break;
      }

      gama = c*c*Thita_; 
      xi = c*c*alpha;    
      normt; 
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_64F,1,&normt,HIP_R_64F,HIP_R_64F)); 
      f = 1 + normt*normt; 
      PSI = gama*(PSI + FI);
      FI = gama*gama*FI + xi*xi*sigma;
      GAMA = GAMA  + 2*PSI + FI;


      DELTA = gama*DELTA - xi*rho_;
      BITA = BITA + DELTA;
      p = ((thita_-ita+2*BITA+GAMA))/f;
      thita = ita+p;

      pk = (((thita_)-ita+BITA)*((thita_)-ita+BITA))/f - p*p;
      rkm = sqrt(((g)*(g))/f + pk);


      if(i==0){
         r00 = rkm;
      }


      if(rho_ < tol && exitLoop == 0){
         break;
      }


      rkm = sqrt(g*g/f);

       if(exitLoop == 0 && ((g < rkm*max(0.99 * sqrt(f),sqrt(g/g_))) || (thita > thita_) || rkm<0.1*r00  || g < tol || rkm < tol)){
         break; 
       }

      hipMemset(w,0,sizeof(double)*dim);
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_64F,1,&scalw,HIP_R_64F,HIP_R_64F)); 
      scalw = 1/scalw;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&scalw,HIP_R_64F,r,HIP_R_64F,1,w,HIP_R_64F,1,HIP_R_64F));

         
      /* rho = r'*w; */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,(void*)r,HIP_R_64F,1,(void*)w,HIP_R_64F,1,(void*)&rho,HIP_R_64F,HIP_R_64F));
      vita = rho/rho_;

      /* d = w + vita*d; */
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&vita,HIP_R_64F,d,HIP_R_64F,1,HIP_R_64F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one,HIP_R_64F,w,HIP_R_64F,1,d,HIP_R_64F,1,HIP_R_64F));


      thita_ = thita;
      Thita_ = Thita;
      rho_ = rho;
      g_ = g;

   }

}

void sqmrD_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);
   hipFree(spsqmr->VTd);

}



void sqmrD_init(double *X, int ldX, double *B, int ldB, double *V,int ldV, int numEvals,
               int dim, double infNormB, struct jdqmr16Info *jd){

   double tmpScalar;
   double minus_one = -1.0;
   double one       =  1.0;
   double zero      =  0.0;



   double *x = X;
   double *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   double *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(double)*dim);
   hipMalloc((void**)&(spsqmr->VTd),sizeof(double)*numEvals);

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)spsqmr->d,HIP_R_64F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)spsqmr->w,HIP_R_64F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_64F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);



   assert(spsqmr->bufferSize>=0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);



}









void sqmr_init(half *X, int ldX, half *B, int ldB, int dim, double infNormB, struct jdqmr16Info *jd){

 
   half tmpScalar;
   half minus_one = __float2half(-1.0);
   half one  = __float2half(1.0);
   half zero = __float2half(0.0);



   half *x = X;
   half *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   half *t = x;
   hipMalloc((void**)&(spsqmr->delta),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->r),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->d),sizeof(half)*dim);
   hipMalloc((void**)&(spsqmr->w),sizeof(half)*dim);

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spsqmr->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesH,
							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_16F);


	hipsparseCreateDnVec(&(spsqmr->descrd),dim,(void*)spsqmr->d,HIP_R_16F);
	hipsparseCreateDnVec(&(spsqmr->descrw),dim,(void*)spsqmr->w,HIP_R_16F);

   hipsparseSpMV_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spsqmr->descrA,spsqmr->descrd,&zero,
                        spsqmr->descrw,HIP_R_16F,HIPSPARSE_COOMV_ALG,&spsqmr->bufferSize);



   assert(spsqmr->bufferSize>=0);
   hipMalloc((void**)&spsqmr->buffer,spsqmr->bufferSize);


}

void sqmr_destroy(struct jdqmr16Info *jd){

   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;


   hipFree(spsqmr->buffer);
   hipFree(spsqmr->w);
   hipFree(spsqmr->delta);
   hipFree(spsqmr->r);
   hipFree(spsqmr->d);

}

void sqmr(half *X, int ldX, half *B, int ldB, int dim, double infNormB, struct jdqmr16Info *jd){

/*
   Solving AX=B with sQMR and early stopping criteria

*/

   // this should be input in this function
   float ita    = 0.0;
   float thita_ = 0.0; 

   int qmrMaxIt  = max(10*dim,1000);
   double tol    = 1e-08;

   
   float Thita_ = 0.0;
   float rho_;
   float sigma;
   float alpha;  
   float normr;
   float Thita;
   float c;
   float g_;
   float g__;
   float r00;
   float rho;
   float vita;
   float g;
   float gama;
   float xi;
   float normt; 
   float f;
   float p;
   float thita;
   float pk;
   float rkm;
   float scalw;
      

   float minus_alpha;
   float deltaScal1;
   float deltaScal2;
   float one32 = 1.0;
      
   float BITA  = 0.0;
   float DELTA = 0.0;
   float GAMA  = 0.0;
   float FI    = 0.0;
   float PSI   = 0.0;


   half tmpScalar;
   half minus_one = __float2half(-1.0);
   half one  = __float2half(1.0);
   half zero = __float2half(0.0);

   float minus_onef = -1.0;
   float onef  = 1.0;
   float zerof = 0.0;

   half *x = X;
   half *b = B;
   
   struct gpuHandler               *gpuH   = jd->gpuH;
   hipblasHandle_t                cublasH   = gpuH->cublasH;
   hipsparseHandle_t              cusparseH = gpuH->cusparseH;
   struct innerSolverSpace       *spInnerSolver = jd->spInnerSolver;         
   struct sqmrSpace              *spsqmr   = spInnerSolver->spSQmr;

   hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);
   hipDeviceSynchronize();


   half *t = x;
   half *delta = (half*)spsqmr->delta;
   half *r = (half*)spsqmr->r;     
   half *d = (half*)spsqmr->d;     
   half *w = (half*)spsqmr->w;     

   /* cusparse data initilization */
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseSpMatDescr_t descrA = spsqmr->descrA;
   hipsparseDnVecDescr_t descrd = spsqmr->descrd;
   hipsparseDnVecDescr_t descrw = spsqmr->descrw;

   size_t bufferSize = spsqmr->bufferSize;// = spsqmr->bufferSize;
   void *buffer = spsqmr->buffer;// = spsqmr->buffer;


   /* r = -b */
   hipMemset(r,0,sizeof(half)*dim);
   hipblasAxpyEx(cublasH,dim,&minus_onef,HIP_R_32F,b,HIP_R_16F,1,r,HIP_R_16F,1,HIP_R_32F);

   /* d = r */
   hipMemset(d,0,sizeof(half)*dim);
   hipblasAxpyEx(cublasH,dim,&onef,HIP_R_32F,r,HIP_R_16F,1,d,HIP_R_16F,1,HIP_R_32F);

   CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,b,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
   g = __half2float(tmpScalar);


   CUBLAS_CALL(hipblasDotEx(cublasH,dim,(void*)r,HIP_R_16F,1,(void*)d,HIP_R_16F,1,(void*)&tmpScalar,HIP_R_16F,HIP_R_32F));
   rho_ = __half2float(tmpScalar);

   /* sQMR main iteration */
   for(int i=0; i<qmrMaxIt; i++){
      jd->innerIterations++;
      
      /* w = A*d */
      hipsparseSpMV(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,
             &one,descrA,descrd,&zero,descrw,HIP_R_16F,
             HIPSPARSE_COOMV_ALG,buffer);
      jd->numMatVecsfp16++;

      /* sigma = d'*w */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,d,HIP_R_16F,1,w,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
      sigma = __half2float(tmpScalar);
      /* alpha = rho_/sigma */
      alpha =1*rho_/sigma;
   

      /* r = r -alpha*w */
      minus_alpha = -1*alpha;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&minus_alpha,HIP_R_32F,w,HIP_R_16F,1,r,HIP_R_16F,1,HIP_R_32F));

      /* Thita = norm(r)/g */
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F));
      normr = __half2float(tmpScalar);
      Thita = normr/g;

      /* c = 1./sqrt(1+Thita*Thita) */
      c = sqrt(1/(1+Thita*Thita));
      /* g = g*Thita*c */
      g__ = g;      
      g =g*Thita*c;
      
      if(i == 0){
         g_ = g;
      }

      /* delta = (c*c*alpha)*d + (c*c*Thita_*Thita_)*delta  */
      deltaScal1 = c*c*Thita_*Thita_;   
      deltaScal2 = c*c*alpha;
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&deltaScal1,HIP_R_32F,delta,HIP_R_16F,1,HIP_R_32F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&deltaScal2,HIP_R_32F,d,HIP_R_16F,1,delta,HIP_R_16F,1,HIP_R_32F));
      /*  t  = t + delta */
      float minus_one32 = -1.0;
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one32,HIP_R_32F,delta,HIP_R_16F,1,t,HIP_R_16F,1,HIP_R_32F));

      if(abs(g) < tol){
         break;
      }

      gama = c*c*Thita_; 
      xi = c*c*alpha;    
      normt; 
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F)); 
      normt = __half2float(tmpScalar);
      f = 1 + normt*normt; 
      PSI = gama*(PSI + FI);
      FI = gama*gama*FI + xi*xi*sigma;
      GAMA = GAMA  + 2*PSI + FI;


      DELTA = gama*DELTA - xi*rho_;
      BITA = BITA + DELTA;
      p = ((thita_-ita+2*BITA+GAMA))/f;
      thita = ita+p;

      pk = (((thita_)-ita+BITA)*((thita_)-ita+BITA))/f - p*p;
      rkm = sqrt(((g)*(g))/f + pk);


      if(i==0){
         r00 = rkm;
      }


      if(rho_ < tol){
         break;
      }


      rkm = sqrt(g*g/f);

       if( (g < rkm*max(0.99 * sqrt(f),sqrt(g/g_))) || (thita > thita_) || rkm<0.1*r00  || g < tol || rkm < tol){
          break; 
       }

      /* w = 0*w+ (1/norm(r))*r */         
      hipMemset(w,0,sizeof(half)*dim);
      CUBLAS_CALL(hipblasNrm2Ex(cublasH,dim,r,HIP_R_16F,1,&tmpScalar,HIP_R_16F,HIP_R_32F)); 
      scalw = 1/__half2float(tmpScalar);
      hipblasAxpyEx(cublasH,dim,&scalw,HIP_R_32F,r,HIP_R_16F,1,w,HIP_R_16F,1,HIP_R_32F);

   


      /* rho = r'*w; */
      CUBLAS_CALL(hipblasDotEx(cublasH,dim,(void*)r,HIP_R_16F,1,(void*)w,HIP_R_16F,1,(void*)&tmpScalar,HIP_R_16F,HIP_R_32F));
      rho = __half2float(tmpScalar);
      vita = rho/rho_;



      /* d = w + vita*d; */
      CUBLAS_CALL(hipblasScalEx(cublasH,dim,&vita,HIP_R_32F,d,HIP_R_16F,1,HIP_R_32F));
      CUBLAS_CALL(hipblasAxpyEx(cublasH,dim,&one32,HIP_R_32F,w,HIP_R_16F,1,d,HIP_R_16F,1,HIP_R_32F));


      thita_ = thita;
      Thita_ = Thita;
      rho_ = rho;
      g_ = g;

   }


}
