#include "../include/helper.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void printMatrixDouble(double *matrix, int rows, int cols, char *name){

	double *hmatrix = (double*)malloc(sizeof(double)*rows*cols);
	hipMemcpy(hmatrix,matrix,sizeof(double)*rows*cols,hipMemcpyDeviceToHost);

	printf("\%% %s\n",name);
	printf("\%% ===================\n");

	 printf("%s = zeros(%d,%d);\n",name,rows,cols);
    for(int row = 0 ; row < rows ; row++){
        for(int col = 0 ; col < cols ; col++){
            double Areg = hmatrix[row + col*rows];
            printf("%s(%d,%d) = %.50lf;\n", name, row+1, col+1, Areg);
        }
    }


	printf("\%% ===================\n\n");
	free(hmatrix);
}

void printMatrixInt(double *matrix, int rows, int cols, char *name){

	int *hmatrix = (int*)malloc(sizeof(int)*rows*cols);
	hipMemcpy(hmatrix,matrix,sizeof(int)*rows*cols,hipMemcpyDeviceToHost);

	printf("\%% %s\n",name);
	printf("\n\n \%% ===================\n");

	 printf("%s = zeros(%d,%d);\n",name,rows,cols);
    for(int row = 0 ; row < rows ; row++){
        for(int col = 0 ; col < cols ; col++){
            int Areg = hmatrix[row + col*rows];
            printf("%s(%d,%d) = %d;\n", name, row+1, col+1, Areg);
        }
    }


	printf("\n\n \%% ===================\n");
	free(hmatrix);

}

