#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hipblas.h>
#include <stdio.h>
#include <assert.h>

#include "initBasis.h"


#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void initBasis_init(double *W, int ldW, double *H, int ldH, double *V, int ldV, double *L,
                int dim, int maxSizeW, int numEvals, struct jdqmr16Info *jd){

   int memReqD    = 0;
   int memReqI    = 0;
   size_t memReqV = 0;
   struct gpuHandler *gpuH = jd->gpuH;   
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   struct initBasisSpace *spInitBasis = jd->spInitBasis;

   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */

   int lwork_geqrf;
   int lwork_orgqr;
   int lwork      ;

   int info_gpu = 0;

   hipMalloc ((void**)&(spInitBasis->d_tau), sizeof(double)*dim);
   hipMalloc ((void**)&(spInitBasis->devInfo), sizeof(int));
   
   hipsolverDnDgeqrf_bufferSize(cusolverH,dim,numEvals,V,ldV,&lwork_geqrf);
   hipsolverDnDorgqr_bufferSize(cusolverH,dim,numEvals,numEvals,V,ldV,spInitBasis->d_tau,&lwork_orgqr);
   spInitBasis->lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
   hipMalloc((void**)&(spInitBasis->d_work), sizeof(double)*(spInitBasis->lwork));
 
   /* allocation of extra space */
   hipMalloc((void**)&spInitBasis->AV,sizeof(double)*dim*numEvals); spInitBasis->ldAV = dim;// AV

   /* cusparse descriptors */
   double one  = 1.0;
   double zero = 0.0;
   hipsparseHandle_t cusparseH = gpuH->cusparseH;
   struct jdqmr16Matrix  *A = jd->matrix;
//   hipsparseCreateCoo(&(spInitBasis->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
//               							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);

   hipsparseCreateCsr(&(spInitBasis->descrA),dim,dim,A->nnz,A->devCsrRows,A->devCols,A->devValuesD,HIPSPARSE_INDEX_32I,
                     HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);


	hipsparseCreateDnMat(&(spInitBasis->descrV),dim,numEvals,dim,V,HIP_R_64F,HIPSPARSE_ORDER_COL);
   hipsparseCreateDnMat(&(spInitBasis->descrAV),dim,numEvals,dim,spInitBasis->AV,HIP_R_64F,HIPSPARSE_ORDER_COL);

	hipDeviceSynchronize();
   size_t bufferSize = -1;
	hipsparseSpMM_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spInitBasis->descrA,spInitBasis->descrV,&zero,spInitBasis->descrAV,
                        HIP_R_64F,HIPSPARSE_SPMM_ALG_DEFAULT,&bufferSize);


   hipMalloc((void**)&(spInitBasis->externalBuffer),bufferSize);
   spInitBasis->bufferSize = bufferSize;    

   memReqV += bufferSize;
   memReqD += dim*numEvals;
   memReqD += spInitBasis->lwork;
   memReqD += dim;
   memReqI += 1;

   jd->gpuMemSpaceDoubleSize = max(jd->gpuMemSpaceDoubleSize,memReqD);
   jd->gpuMemSpaceIntSize    = max(jd->gpuMemSpaceIntSize,memReqI);
   jd->gpuMemSpaceVoidSize   = max(jd->gpuMemSpaceVoidSize,memReqV);

#if 0
   hipFree(spInitBasis->d_tau);
   hipFree(spInitBasis->devInfo);
   hipFree(spInitBasis->d_work);
   hipFree(spInitBasis->externalBuffer);
   hipFree(spInitBasis->AV);
#endif
}


void initBasis_destroy(struct jdqmr16Info *jd){

   struct initBasisSpace *spInitBasis = jd->spInitBasis;

#if 1
   hipFree(spInitBasis->externalBuffer);
   hipFree(spInitBasis->d_tau);
   hipFree(spInitBasis->devInfo);
   hipFree(spInitBasis->d_work);
   hipFree(spInitBasis->AV);
#endif
}

void initBasis(double *W, int ldW, double *H, int ldH, double *V, int ldV, double *L, double *AW, int ldAW,
                int dim, int maxSizeW, int numEvals, int seed, struct jdqmr16Info *jd){


   struct gpuHandler *gpuH = jd->gpuH;   
   /* Step 1: Random initialization of V */

   if(seed == 1){
      hiprandGenerator_t curandH = gpuH->curandH;
	   hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */

	   double  mean = 0.0;
	   double  stddev = max(dim,numEvals);

      hipMemset(V,0,dim*numEvals);
	   hiprandGenerateNormalDouble(curandH, V, dim*numEvals,mean,stddev); /* Generate dim*maxSizeW on device */
   }
   /* Step 2: Orthogonalization of V */
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   struct initBasisSpace *spInitBasis = jd->spInitBasis;

#if 0

   double *memD = jd->gpuMemSpaceDouble;
   int    *memI = jd->gpuMemSpaceInt;
   void   *memV = jd->gpuMemSpaceVoid;

   double *d_tau  = memD; memD+= dim;
   double *AV     = memD; memD+= dim;// spInitBasis->AV; 
   int ldAV = spInitBasis->ldAV;

   double *d_work = memD;
   void   *buffer = memV;
   int    *devInfo = memI;

   hipsparseCreateDnMat(&(spInitBasis->descrAV),dim,numEvals,dim,AV,HIP_R_64F,HIPSPARSE_ORDER_COL);

#else

   double *d_tau  = spInitBasis->d_tau;
   double *d_work = spInitBasis->d_work;
   void   *buffer = spInitBasis->externalBuffer;
   int    *devInfo = spInitBasis->devInfo;
   double *AV = spInitBasis->AV; 
   int ldAV = spInitBasis->ldAV;
 
#endif



   int info_gpu = 0;

   hipsolverDnDgeqrf(cusolverH,dim,numEvals,V,ldV,d_tau,d_work,spInitBasis->lwork,devInfo);
   hipsolverDnDorgqr(cusolverH,dim,numEvals,numEvals,V,ldV,d_tau,d_work,spInitBasis->lwork,devInfo);


   /* Projection of A into V : H = V'AV  */
   struct devSolverSpace *sp = jd->sp;
   hipsparseHandle_t cusparseH = gpuH->cusparseH;
   hipMemset(H,0,maxSizeW*maxSizeW*sizeof(double));
   //double *AV = spInitBasis->AV; 
   hipMemset(AV,0,dim*numEvals*sizeof(double));
  
   struct jdqmr16Matrix  *A = jd->matrix;

	assert(spInitBasis->descrA != NULL || spInitBasis->descrV != NULL || spInitBasis->descrAV != NULL);
	hipDeviceSynchronize();


   double one  = 1.0;
	double zero = 0.0;

	size_t bufferSize = -1;

   hipsparseSpMM(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,&one,
             spInitBasis->descrA,spInitBasis->descrV,&zero,spInitBasis->descrAV,
             HIP_R_64F,HIPSPARSE_SPMM_ALG_DEFAULT,buffer);
   jd->numMatVecsfp64++;

   hipMemcpy(AW,AV,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);

   hipblasHandle_t cublasH = gpuH->cublasH;

   hipblasDgemm(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals, dim,&one,V,ldV,AV,ldAV,&zero,H,ldH);

   /* W = V */
   hipMemcpy(W,V,dim*numEvals*sizeof(double),hipMemcpyDeviceToDevice); 

}


















