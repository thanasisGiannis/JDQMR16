#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hipblas.h>
#include <stdio.h>
#include <assert.h>

#include "initBasis.h"


#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void initBasis_init(double *W, int ldW, double *H, int ldH, double *V, int ldV, double *L,
                int dim, int maxSizeW, int numEvals, struct jdqmr16Info *jd){

   struct gpuHandler *gpuH = jd->gpuH;   
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   struct initBasisSpace *spInitBasis = jd->spInitBasis;

   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */

   int lwork_geqrf;
   int lwork_orgqr;
   int lwork      ;

   int info_gpu = 0;

   hipMalloc ((void**)&(spInitBasis->d_tau), sizeof(double)*dim);
   hipMalloc ((void**)&(spInitBasis->devInfo), sizeof(int));
   hipMalloc ((void**)&(spInitBasis->d_R ) , sizeof(double)*numEvals*numEvals);

   hipsolverDnDgeqrf_bufferSize(cusolverH,dim,numEvals,V,ldV,&lwork_geqrf);
   hipsolverDnDorgqr_bufferSize(cusolverH,dim,numEvals,numEvals,V,ldV,spInitBasis->d_tau,&lwork_orgqr);
   spInitBasis->lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
   hipMalloc((void**)&(spInitBasis->d_work), sizeof(double)*(spInitBasis->lwork));


   /* allocation of extra space */
   hipMalloc((void**)&spInitBasis->AV,sizeof(double)*dim*numEvals); spInitBasis->ldAV = dim;// AV


   /* cusparse descriptors */
   double one  = 1.0;
   double zero = 0.0;
   hipsparseHandle_t cusparseH = gpuH->cusparseH;
   struct jdqmr16Matrix  *A = jd->matrix;
   hipsparseCreateCoo(&(spInitBasis->descrA),dim,dim,A->nnz,A->devRows,A->devCols,A->devValuesD,
               							HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO,HIP_R_64F);

	hipsparseCreateDnMat(&(spInitBasis->descrV),dim,numEvals,dim,V,HIP_R_64F,HIPSPARSE_ORDER_COL);

   hipsparseCreateDnMat(&(spInitBasis->descrAV),dim,numEvals,dim,spInitBasis->AV,HIP_R_64F,HIPSPARSE_ORDER_COL);
	hipDeviceSynchronize();
   size_t bufferSize = -1;
	hipsparseSpMM_bufferSize(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &one,spInitBasis->descrA,spInitBasis->descrV,&zero,spInitBasis->descrAV,
                        HIP_R_64F,HIPSPARSE_COOMM_ALG2,&bufferSize);


   hipMalloc((void**)&(spInitBasis->externalBuffer),bufferSize);
   spInitBasis->bufferSize = bufferSize;    

}


void initBasis_destroy(struct jdqmr16Info *jd){

   struct initBasisSpace *spInitBasis = jd->spInitBasis;

   hipFree(spInitBasis->externalBuffer);
   hipFree(spInitBasis->d_tau);
   hipFree(spInitBasis->devInfo);
   hipFree(spInitBasis->d_R);
   hipFree(spInitBasis->d_work);
   hipFree(spInitBasis->AV);
}

void initBasis(double *W, int ldW, double *H, int ldH, double *V, int ldV, double *L, double *AW, int ldAW,
                int dim, int maxSizeW, int numEvals, struct jdqmr16Info *jd){


   struct gpuHandler *gpuH = jd->gpuH;   
   /* Step 1: Random initialization of V */
   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */

	double  mean = 0.0;
	double  stddev = max(dim,numEvals);

   hipMemset(V,0,dim*numEvals);
	hiprandGenerateNormalDouble(curandH, V, dim*numEvals,mean,stddev); /* Generate dim*maxSizeW on device */

   /* Step 2: Orthogonalization of V */
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   struct initBasisSpace *spInitBasis = jd->spInitBasis;

   double *d_tau = spInitBasis->d_tau;
   int    *devInfo = spInitBasis->devInfo;
   double *d_work = spInitBasis->d_work;

   double *d_R = NULL;
   int info_gpu = 0;

   hipsolverDnDgeqrf(cusolverH,dim,numEvals,V,ldV,d_tau,d_work,spInitBasis->lwork,devInfo);
   hipsolverDnDorgqr(cusolverH,dim,numEvals,numEvals,V,ldV,d_tau,d_work,spInitBasis->lwork,devInfo);


   /* Projection of A into V : H = V'AV  */
   struct devSolverSpace *sp = jd->sp;
   hipsparseHandle_t cusparseH = gpuH->cusparseH;
   hipMemset(H,0,maxSizeW*maxSizeW*sizeof(double));
   double *AV = spInitBasis->AV; hipMemset(AV,0,dim*numEvals*sizeof(double));
   int ldAV = spInitBasis->ldAV;

   struct jdqmr16Matrix  *A = jd->matrix;

	assert(spInitBasis->descrA != NULL || spInitBasis->descrV != NULL || spInitBasis->descrAV != NULL);
	hipDeviceSynchronize();


   double one  = 1.0;
	double zero = 0.0;

	size_t bufferSize = -1;

   hipsparseSpMM(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,&one,
             spInitBasis->descrA,spInitBasis->descrV,&zero,spInitBasis->descrAV,
             HIP_R_64F,HIPSPARSE_COOMM_ALG2,spInitBasis->externalBuffer);


   hipMemcpy(AW,AV,sizeof(double)*dim*numEvals,hipMemcpyDeviceToDevice);

   hipblasHandle_t cublasH = gpuH->cublasH;

   hipblasDgemm(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numEvals,numEvals, dim,&one,V,ldV,AV,ldAV,&zero,H,ldH);

   /* W = V */
   hipMemcpy(W,V,dim*numEvals*sizeof(double),hipMemcpyDeviceToDevice); 

}


















