#include "hip/hip_runtime.h"
#include <hiprand.h>
#include "initBasis.h"
#include <stdio.h>
#include "../include/helper.h"
#include "../../include/jdqmr16.h"

void initBasis_init(double *W, int ldW, double *H, int ldH, double *V, int ldV, double *L,
                int dim, int maxSizeW, int numEvals, struct jdqmr16Info *jd){

   struct gpuHandler *gpuH = jd->gpuH;   
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   struct initBasisSpace *spInitBasis = jd->spInitBasis;

   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */

   int lwork_geqrf;
   int lwork_orgqr;
   int lwork      ;

   int info_gpu = 0;

   hipMalloc ((void**)&(spInitBasis->d_tau), sizeof(double)*dim);
   hipMalloc ((void**)&(spInitBasis->devInfo), sizeof(int));
   hipMalloc ((void**)&(spInitBasis->d_R ) , sizeof(double)*numEvals*numEvals);

   hipsolverDnDgeqrf_bufferSize(cusolverH,dim,numEvals,V,ldV,&lwork_geqrf);
   hipsolverDnDorgqr_bufferSize(cusolverH,dim,numEvals,numEvals,V,ldV,spInitBasis->d_tau,&lwork_orgqr);
   spInitBasis->lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
   hipMalloc((void**)&(spInitBasis->d_work), sizeof(double)*(spInitBasis->lwork));

}


void initBasis_destroy(struct jdqmr16Info *jd){

   struct initBasisSpace *spInitBasis = jd->spInitBasis;
   hipFree(spInitBasis->d_tau);
   hipFree(spInitBasis->devInfo);
   hipFree(spInitBasis->d_R);
   hipFree(spInitBasis->d_work);

}

void initBasis(double *W, int ldW, double *H, int ldH, double *V, int ldV, double *L,
                int dim, int maxSizeW, int numEvals, struct jdqmr16Info *jd){


   struct gpuHandler *gpuH = jd->gpuH;   
   /* Step 1: Random initialization of V */
   hiprandGenerator_t curandH = gpuH->curandH;
	hiprandSetPseudoRandomGeneratorSeed(curandH,1234ULL); /* set Seed */

	double  mean = 0.0;
	double  stddev = max(dim,numEvals);

   hipMemset((void**)V,0,dim*numEvals);
	hiprandGenerateNormalDouble(curandH, V, dim*numEvals,mean,stddev); /* Generate dim*maxSizeW on device */

   /* Step 2: Orthogonalization of V */
   hipsolverHandle_t cusolverH = gpuH->cusolverH;
   struct initBasisSpace *spInitBasis = jd->spInitBasis;

   double *d_tau = spInitBasis->d_tau;
   int    *devInfo = spInitBasis->devInfo;
   double *d_work = spInitBasis->d_work;

   double *d_R = NULL;
   int info_gpu = 0;

   hipsolverDnDgeqrf(cusolverH,dim,numEvals,V,ldV,d_tau,d_work,spInitBasis->lwork,devInfo);
   hipsolverDnDorgqr(cusolverH,dim,numEvals,numEvals,V,ldV,d_tau,d_work,spInitBasis->lwork,devInfo);
   printMatrixDouble(V,dim,numEvals,"V");
}
